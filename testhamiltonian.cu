#include "hip/hip_runtime.h"
#include"testhamiltonian.h"

/* NOTE: this function uses FORTRAN style matrices, where the values and positions are stored in a ONE dimensional array! Don't forget this! */

/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
	lattice_Size - the number of sites
	Sz - the value of the Sz operator
	basis_Position[] - an empty array that records the positions of the basis
	basis - an empty array that records the basis
Outputs:	basis_Position - a full array now
		basis[] - a full array now

*/
__host__ long GetBasis(long dim, int lattice_Size, int Sz, long basis_Position[], long basis[]){
	unsigned long temp = 0;
	long realdim = 0;

	for (unsigned long i1=0; i1<dim; i1++){
      		temp = 0;
		basis_Position[i1] = -1;
      		for (int sp =0; sp<lattice_Size; sp++){
          		temp += (i1>>sp)&1;
		}  //unpack bra
      		if (temp==(lattice_Size/2+Sz) ){ 
          		basis[realdim] = i1;
          		basis_Position[i1] = realdim;
			realdim++;
			//cout<<basis[realdim]<<" "<<basis_Position[i1]<<endl;
      		}
  	} 

	return realdim;

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
        bra - the state
        JJ - the coupling constant
Outputs:  valH - the value of the Hamiltonian 

*/

__device__ hipDoubleComplex HOffBondX(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0.); //contribution from the J part of the Hamiltonian

  	return valH;


} 

__device__ hipDoubleComplex HOffBondY(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0. ); //contribution from the J part of the Hamiltonian

  	return valH;


}

__device__ hipDoubleComplex HDiagPart(const long bra, int lattice_Size, long3* d_Bond, const double JJ){

  int S0b,S1b ;  //spins (bra 
  int T0,T1;  //site
  int P0, P1, P2, P3; //sites for plaquette (Q)
  int s0p, s1p, s2p, s3p;
  hipDoubleComplex valH = make_hipDoubleComplex( 0. , 0.);

  for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

    T0 = (d_Bond[Ti]).x; //lower left spin
    S0b = (bra>>T0)&1;  
    //if (T0 != Ti) cout<<"Square error 3\n";
    T1 = (d_Bond[Ti]).y; //first bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);
    T1 = (d_Bond[Ti]).z; //second bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);

  }//T0

  //cout<<bra<<" "<<valH<<endl;

  return valH;

}//HdiagPart 

/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
	lattice_Size - the number of lattice sites
	Bond - the bond values ??
	hamil_Values - an empty pointer for a device array containing the values 
	hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
	hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs:  hamil_Values - a pointer to a device array containing the values 
	hamil_PosRow - a pointer to a device array containing the locations of each value in a row
	hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


__host__ int ConstructSparseMatrix(int model_Type, int lattice_Size, long* Bond, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol){


        std::ofstream fout;
        fout.open("testhamiltonian.log"); //creating a log file to store timing information

	long num_Elem = 0; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	long dim = 65536;
	long vdim;
	/*
	switch (model_Type){
		case 0: dim = 65536;
		case 1: dim = 10; //guesses
	}
        */
      
	//for (int ch=1; ch<lattice_Size; ch++) dim *= 2;

	int stridepos = 2*lattice_Size + 2;
	int strideval = 2*lattice_Size + 1;        

	long basis_Position[dim];
	long basis[dim];

	int Sz = 0;

	//----------------Construct basis and copy it to the GPU --------------------//
	hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start,0);
        vdim = GetBasis(dim, lattice_Size, Sz, basis_Position, basis);
        hipEventRecord(stop,0);

        float elapsed;

        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Run time for GetBasis: "<<elapsed<<std::endl;

	long* d_basis_Position;
	long* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(long));
	status2 = hipMalloc(&d_basis, vdim*sizeof(long));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory allocation for basis arrays failed! Error: ";
                std::cout<<hipPeekAtLastError()<<std::endl;
		return 1;
	}

	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(long), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, vdim*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory copy for basis arrays failed! Error: ";
                std::cout<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	long* d_Bond;
	status1 = hipMalloc(&d_Bond, 3*lattice_Size*sizeof(long));

	status2 = hipMemcpy(d_Bond, Bond, 3*lattice_Size*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory allocation and copy for bond data failed! Error: ";
                std::cout<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}	

	dim3 bpg;

	if (vdim <= 65336);
                bpg.x = vdim;
        
	dim3 tpb;
        tpb.x = 32;
        //these are going to need to depend on dim and Nsize

	//--------------Declare the Hamiltonian arrays on the device, and copy the pointers to them to the device -----------//
	
        long2* d_H_pos;
        hipDoubleComplex* d_H_vals;

        status1 = hipMalloc(&d_H_pos, vdim*stridepos*sizeof(long2));
        status2 = hipMalloc(&d_H_vals, vdim*strideval*sizeof(hipDoubleComplex));

        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
              std::cout<<"Memory allocation for device Hamiltonian failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
              return 1;
        }

        //the above code should work on devices of any compute capability - YEAAAAH

	// --------------------- Fill up the sparse matrix and compress it to remove extraneous elements ------//
  
        double JJ = 1.;

        //std::cout<<"Running FillSparse"<<std::endl;
      
        hipEventRecord(start, 0);
	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, vdim, d_H_vals, d_H_pos, d_Bond, lattice_Size, JJ);

        if( hipPeekAtLastError() != 0 ){
		std::cout<<"Error in FillSparse! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}
		
	hipDeviceSynchronize(); //need to make sure all elements are initialized before I start compression
        hipEventRecord(stop, 0);
        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Runtime for FillSparse: "<<elapsed<<std::endl;

        long* num_ptr;

        hipEventRecord(start,0);

        status1 = hipGetSymbolAddress((void**)&num_ptr, (const char *)"d_num_Elem");
        status2 = hipMemset(num_ptr, 0, sizeof(long));

        hipEventRecord(stop,0);
        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Time to get and set d_num_Elem: "<<elapsed<<std::endl;
        
        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
              std::cout<<"Getting and setting d_num_Elem failed! Error: "<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
              return 1;
        }

	status1 = hipFree(d_basis);
        status2 = hipFree(d_basis_Position);
        status3 = hipFree(d_Bond); // we don't need these later on
	
        if ( (status1 != hipSuccess) || 
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ){
          std::cout<<"Freeing bond and basis information failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
          return 1;
        }

        //std::cout<<"Running CompressSparse"<<std::endl;

        hamstruct* d_H_sort;
        status2 = hipMalloc(&d_H_sort, vdim*strideval*sizeof(hamstruct));

	if (status2 != hipSuccess){
                std::cout<<"Allocating d_H_sort failed! Error: ";
                std::cout<<hipGetErrorString( status1 )<<std::endl;
                return 1;
        }

        GetNumElem<<<vdim/512 + 1, 512>>>(d_H_pos, lattice_Size);
        hipDeviceSynchronize();

        hipEventRecord(start, 0);	
	CompressSparse<<<vdim, 32>>>(d_H_vals, d_H_pos, d_H_sort, vdim, lattice_Size);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Runtime for CompressSparse: "<<elapsed<<std::endl;

        if (hipPeekAtLastError() != 0){
              std::cout<<"Error in CompressSparse! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
              return 1;
        }

	hipFree(d_H_vals); //cleanup
	hipFree(d_H_pos);

	status1 = hipMemcpy(&num_Elem, num_ptr, sizeof(long), hipMemcpyDeviceToHost);
	num_Elem = 2*num_Elem - vdim;

        std::cout<<"Number of nonzero elements: "<<num_Elem<<std::endl;

        if (status1 != hipSuccess){
              std::cout<<"Copying number of elements failed! Error: "<<hipGetErrorString(status1)<<std::endl;
              return 1;
	}
        //----------------Sorting Hamiltonian--------------------------//

        thrust::device_ptr<hamstruct> sort_ptr(d_H_sort);

        hipEventRecord(start,0);
        thrust::sort(sort_ptr, sort_ptr + num_Elem, ham_sort_function());
        
        //--------------------------------------------------------------

        hipDeviceSynchronize();
        hipEventRecord(stop,0);
        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Runtime for sorting: "<<elapsed<<std::endl;

        if (hipPeekAtLastError() != 0){
                std::cout<<"Error in sorting! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
                return 1;
        }

        //std::cout<<"Sorting complete"<<std::endl;

        
	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&hamil_PosRow, num_Elem*sizeof(long));
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(long));

	if ( (status1 != hipSuccess) ||
	     (status2 != hipSuccess) ||
	     (status3 != hipSuccess) ){
		std::cout<<"Memory allocation for COO representation failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}
        
				
	//std::cout<<"Running FullToCOO."<<std::endl;

        hipEventRecord(start, 0);	
        FullToCOO<<<num_Elem/512 + 1, 512>>>(num_Elem, d_H_sort, hamil_Values, hamil_PosRow, hamil_PosCol, vdim); // csr and description initializations happen somewhere else

        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventElapsedTime(&elapsed, start, stop);

        fout<<"Runtime for FullToCOO: "<<elapsed<<std::endl;
	
	hipFree(d_H_sort);

        hipEventDestroy(start);
        hipEventDestroy(stop);
       
        fout.close();
	return 0;
}

int main(){
        //hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024); //have to set a heap size or malloc()s on the device will fail
        
	long* Bond;
	Bond = (long*)malloc(16*3*sizeof(long));
	
	Bond[0] = 0; 	Bond[1] = 1; 	Bond[2] = 2;	Bond[3] = 3;	Bond[4] = 4;	Bond[5] = 5;
	Bond[6] = 6; 	Bond[7] = 7;	Bond[8] = 8;	Bond[9] = 9;	Bond[10] = 10;	Bond[11] = 11;
	Bond[12] = 12;	Bond[13] = 13;	Bond[14] = 14;	Bond[15] = 15;	Bond[16] = 1; 	Bond[17] = 2;
	Bond[18] = 3; 	Bond[19] = 0;	Bond[20] = 5; 	Bond[21] = 6;	Bond[22] = 7;	Bond[23] = 4;
	Bond[24] = 9;	Bond[25] = 10; 	Bond[26] = 11;	Bond[27] = 8;	Bond[28] = 13;	Bond[29] = 14;
	Bond[30] = 15; 	Bond[31] = 12; 	Bond[32] = 4;	Bond[33] = 5;	Bond[34] = 6;	Bond[35] = 7;
	Bond[36] = 8;	Bond[37] = 9;	Bond[38] = 10;	Bond[39] = 11;	Bond[40] = 12;	Bond[41] = 13;
	Bond[42] = 14;	Bond[43] = 15;	Bond[44] = 0;	Bond[45] = 1;	Bond[46] = 2;	Bond[47] = 3;

	hipDoubleComplex* hamil_Values;

	long* hamil_PosRow;

	long* hamil_PosCol;


	int rtn = ConstructSparseMatrix(0, 16, Bond, hamil_Values, hamil_PosRow, hamil_PosCol);

	free(Bond);
        hipFree(hamil_Values);
        hipFree(hamil_PosRow);
        hipFree(hamil_PosCol);
	return rtn;
}
/* Function FillSparse: this function takes the empty Hamiltonian arrays and fills them up. Each thread in x handles one ket |i>, and each thread in y handles one site T0
Inputs: d_basis_Position - position information about the basis
	d_basis - other basis infos
	d_dim - the number of kets
	H_vals - an array that will store the values
	H_pos - an array that will store the positions of things
	d_Bond - the bond information
	d_lattice_Size - the number of lattice sites
	JJ - the coupling parameter 

*/

__global__ void FillSparse(long* d_basis_Position, long* d_basis, int dim, hipDoubleComplex* H_vals, long2* H_pos, long* d_Bond, int lattice_Size, const double JJ){

	
	long ii = blockIdx.x;
        int T0 = threadIdx.x;

        __shared__ long3 tempbond[16];
        __shared__ int count;
	atomicExch(&count, 0);
        __shared__ long temppos[32];
        __shared__ hipDoubleComplex tempval[32]; //going to eliminate a huge number of read/writes to d_Bond, H_vals, H_pos in global memory

        int stridepos = 2*lattice_Size + 2;
        int strideval = 2*lattice_Size + 1;


	int si, sj,sk,sl; //spin operators
	unsigned long tempi, tempj, tempod;
	hipDoubleComplex tempD;

        tempi = d_basis[ii];

	__syncthreads();

	if( ii < dim ){
            if (T0 < lattice_Size){
    		
		//Putting bond info in shared memory
		(tempbond[T0]).x = d_Bond[T0];
		(tempbond[T0]).y = d_Bond[lattice_Size + T0];
		(tempbond[T0]).z = d_Bond[2*lattice_Size + T0];
		
		//Diagonal Part

		temppos[0] = d_basis_Position[tempi];

		tempval[0] = HDiagPart(tempi, lattice_Size, tempbond, JJ);

                H_vals[ idx(ii, 0, strideval) ] = tempval[0];
                (H_pos[ idx(ii, 1, stridepos) ]).y = temppos[0];
                (H_pos[ idx(ii, 1, stridepos) ]).x = ii;
                
                

		//-------------------------------
		//Horizontal bond ---------------
		si = (tempbond[T0]).x;
		tempod = tempi;
		sj = (tempbond[T0]).y;
	
		tempod ^= (1<<si);   //toggle bit 
		tempod ^= (1<<sj);   //toggle bit 

		if (d_basis_Position[tempod] > ii){ //build only upper half of matrix
        		temppos[2*T0] = d_basis_Position[tempod];
        		tempval[2*T0] = HOffBondX(T0,tempi, JJ);
                        atomicAdd(&count,1);
                                                                        
      		}

		else {
			temppos[2*T0] = -1;
			tempval[2*T0] = make_hipDoubleComplex(0., 0.);
		}

		//Vertical bond -----------------
		tempod = tempi;
      		sj = (tempbond[T0]).z;

      		tempod ^= (1<<si);   //toggle bit 
     		tempod ^= (1<<sj);   //toggle bit
                 
      		if (d_basis_Position[tempod] > ii){ 
        		temppos[2*T0 + 1] = d_basis_Position[tempod];
        		tempval[2*T0 + 1] = HOffBondY(T0,tempi, JJ);
			atomicAdd(&count,1);           
      		}

		else {
			temppos[2*T0 + 1] = -1;
			tempval[2*T0 + 1] = make_hipDoubleComplex(0., 0.);
		}

                //time to write back to global memory
                __syncthreads;

		
                H_vals[ idx(ii, 2*T0, strideval) ] = tempval[2*T0];
                (H_pos[ idx(ii, 2*T0, stridepos) ]).y = temppos[2*T0];
                (H_pos[ idx(ii, 2*T0, stridepos) ]).x = ii;                

		(H_pos[ idx(ii, 0, stridepos) ]).x = ii;
                (H_pos[ idx(ii, 0, stridepos) ]).y = count + 1;
                               
                (H_pos[ idx(ii, 2*T0 + 1, stridepos) ]).x = ii;
                (H_pos[ idx(ii, 2*T0 + 1, stridepos) ]).y = temppos[2*T0 + 1];
                
                H_vals[ idx(ii, 2*T0 + 1, strideval) ] = tempval[2*T0 + 1];
                                                 
            }//end of T0 

	}//end of ii
}//end of FillSparse

/* Function: CompressSparse - this function takes the sparse matrix with lots of "buffer" memory sitting on the end of each array, and compresses it down to get rid of the extra memory
Parameters:	H_vals - an array of Hamiltonian values
	        H_pos - an array of value positions 
                H_sort - an array of hamstructs that is filled with the upper and lower halves of the Hamiltonian
	        d_dim - the dimension of the Hamiltonian
	        lattice_Size - the number of lattice sites

*/
__global__ void CompressSparse(const hipDoubleComplex* H_vals, const long2* H_pos, hamstruct* H_sort, long d_dim, const int lattice_Size){

	long row = blockIdx.x;
        int col = threadIdx.x;

        __shared__ long s_H_pos[32];
        __shared__ hipDoubleComplex s_H_vals[32];
        //__shared__ hamstruct s_H_sort[32];

        __shared__ int iter;
        iter = 1;

	const int size1 = 2*lattice_Size + 2;
       	const int size2 = 2*lattice_Size + 1;

        __syncthreads();

        if (row < d_dim){
        
                // the basic idea here is to have each x thread go to the ith row, and each y thread go to the jth element of that row. then using a set of __shared__ temp arrays, we read in the Hamiltonian values and do our comparisons n stuff on them

		long start = 0;
	
                for (long ii = 0; ii < row; ii++){ 
                        start += 2*(H_pos[ idx(ii, 0 , size1) ]).y - 1 ;
        	}
		
		(H_sort[ start ]).rowindex = row;
                (H_sort[ start ]).colindex = row;
                (H_sort[ start ]).value = H_vals[ idx( row, 0, size2) ];
                (H_sort[ start ]).dim = d_dim; //doing the diagonals


		int temp;

                if (col < size2  - 1){
                        s_H_pos[col] = (H_pos[ idx( row, col + 2, size1 ) ]).y;
			s_H_vals[col] = H_vals[ idx( row, col + 1, size2) ];

                	if (s_H_pos[col] != -1){
				temp = atomicAdd(&iter, 1);
				(H_sort[start + temp]).rowindex = row;
				(H_sort[start + temp]).colindex = s_H_pos[col];
				(H_sort[start + temp]).value = s_H_vals[col];
				(H_sort[start + temp]).dim = d_dim;

				temp = atomicAdd(&iter, 1);

				(H_sort[ start + temp ]).rowindex = s_H_pos[col]; //the conjugate
                                (H_sort[ start + temp ]).colindex = row;
                                (H_sort[ start + temp ]).value = make_hipDoubleComplex( (s_H_vals[col]).x , -( s_H_vals[col]).y );
                                (H_sort[ start + temp ]).dim = d_dim;                 
			
			} 
		}

	}
}

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
	 H_vals - the Hamiltonian values
	 H_pos - the Hamiltonian positions
	 hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(long num_Elem, hamstruct* H_sort, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol, long dim){

	int i = threadIdx.x + blockDim.x*blockIdx.x;

	long start = 0;

	if (i < num_Elem){
			
		hamil_Values[i] = H_sort[i].value;
		hamil_PosRow[i] = H_sort[i].rowindex;
		hamil_PosCol[i] = H_sort[i].colindex;
		
	}
}

__global__ void GetNumElem(long2* H_pos, int lattice_Size){

	long row = blockIdx.x*blockDim.x + threadIdx.x;

	atomicAdd(&d_num_Elem, (H_pos[ idx(row, 0, 2*lattice_Size + 2) ]).y);
}
