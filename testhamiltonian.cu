#include "hip/hip_runtime.h"
#include"testhamiltonian.h"

/* NOTE: this function uses FORTRAN style matrices, where the values and positions are stored in a ONE dimensional array! Don't forget this! */

/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
	lattice_Size - the number of sites
	Sz - the value of the Sz operator
	basis_Position[] - an empty array that records the positions of the basis
	basis - an empty array that records the basis
Outputs:	basis_Position - a full array now
		basis[] - a full array now

*/
__host__ int GetBasis(long dim, int lattice_Size, int Sz, long basis_Position[], long basis[]){
	unsigned long temp = 0;
	long realdim = 0;

	long iter = 0;

	for (unsigned long i1=0; i1<dim; i1++){
      		temp = 0;
		basis_Position[i1] = -1;
      		for (int sp =0; sp<lattice_Size; sp++){
          		temp += (i1>>sp)&1;
		}  //unpack bra
      		if (temp==(lattice_Size/2+Sz) ){ 
          		basis[iter] = i1;
          		basis_Position[i1] = i1 -1;
			realdim++;
			iter++;
      		}
  	} 

	return realdim;

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
        bra - the state
        JJ - the coupling constant
Outputs:  valH - the value of the Hamiltonian 

*/

__device__ hipDoubleComplex HOffBondX(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0.); //contribution from the J part of the Hamiltonian

  	return valH;


} 

__device__ hipDoubleComplex HOffBondY(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0. ); //contribution from the J part of the Hamiltonian

  	return valH;


}

__device__ hipDoubleComplex HDiagPart(const long bra, int lattice_Size, long3* d_Bond, const double JJ){

  int S0b,S1b ;  //spins (bra 
  int T0,T1;  //site
  int P0, P1, P2, P3; //sites for plaquette (Q)
  int s0p, s1p, s2p, s3p;
  hipDoubleComplex valH = make_hipDoubleComplex( 0. , 0.);

  for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

    T0 = (d_Bond[Ti]).x; //lower left spin
    S0b = (bra>>T0)&1;  
    //if (T0 != Ti) cout<<"Square error 3\n";
    T1 = (d_Bond[Ti]).y; //first bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);
    T1 = (d_Bond[Ti]).z; //second bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);

  }//T0

  //cout<<bra<<" "<<valH<<endl;

  return valH;

}//HdiagPart 

/* Function: SetFirst - sets the first row elements of a matrix to some value

Inputs: H_pos - an array on the device whose first row elements will be changed
	dim - the number of rows in H_pos
	value - the value we want to set the first elements to
*/

__global__ void SetFirst(long2* H_pos, long stride, long dim, long value){
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i < dim){
      
      (H_pos[ idx(i, 0, stride) ]).y = value;
    }
}

__device__ int nthdigit(long x, int n){
      return (x/10^n)%10;
}

/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
	lattice_Size - the number of lattice sites
	Bond - the bond values ??
	hamil_Values - an empty pointer for a device array containing the values 
	hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
	hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs:  hamil_Values - a pointer to a device array containing the values 
	hamil_PosRow - a pointer to a device array containing the locations of each value in a row
	hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


__host__ int ConstructSparseMatrix(int model_Type, int lattice_Size, long* Bond, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol){
	
	unsigned long num_Elem = 0; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	long dim = 65536;
	long vdim;
	/*
	switch (model_Type){
		case 0: dim = 65536;
		case 1: dim = 10; //guesses
	}
        */
      
	//for (int ch=1; ch<lattice_Size; ch++) dim *= 2;

        status1 = hipSetDeviceFlags(hipDeviceMapHost);

	int stridepos = 2*lattice_Size + 2;
	int strideval = 2*lattice_Size + 1;        

	long basis_Position[dim];
	long basis[dim];

	int Sz = 0;

	//----------------Construct basis and copy it to the GPU --------------------//
	vdim = GetBasis(dim, lattice_Size, Sz, basis_Position, basis);

	cout<<dim<<endl;

	long* d_basis_Position;
	long* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(long));
	status2 = hipMalloc(&d_basis, vdim*sizeof(long));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation for basis arrays failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}


	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(long), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, vdim*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory copy for basis arrays failed! Error: ";
                cout<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}

	long* d_Bond;
	status1 = hipMalloc(&d_Bond, 3*lattice_Size*sizeof(long));

	status2 = hipMemcpy(d_Bond, Bond, 3*lattice_Size*sizeof(long), hipMemcpyHostToDevice);

	status1 = hipPeekAtLastError();

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation and copy for bond data failed! ";
		return 1;
	}	


	dim3 bpg;
        bpg.x = vdim/64;
        bpg.y = lattice_Size;
	dim3 tpb;
        tpb.x = 64;
        tpb.y = 16; //these are going to need to depend on dim and Nsize

	//--------------Declare the Hamiltonian arrays on the device, and copy the pointers to them to the device -----------//

        cout<<vdim<<endl; 	

	long2* h_H_pos;
	hipDoubleComplex* h_H_vals; 

	h_H_pos = (long2*)malloc(vdim*stridepos*sizeof(long2));
        h_H_vals = (hipDoubleComplex*)malloc(vdim*strideval*sizeof(hipDoubleComplex));

	
        long2* d_H_pos;
        hipDoubleComplex* d_H_vals;

        status1 = hipMalloc(&d_H_pos, vdim*stridepos*sizeof(long2));
        status2 = hipMalloc(&d_H_vals, vdim*strideval*sizeof(hipDoubleComplex));

        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
              cout<<"Memory allocation for device Hamiltonian failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
              return 1;
        }

        //the above code should work on devices of any compute capability - YEAAAAH

        cout<<"Running SetFirst"<<endl;	
     
        SetFirst<<<256, 256>>>(d_H_pos, stridepos, vdim, 1); //count the diagonal element
        cout<<hipGetErrorString( hipPeekAtLastError() )<<endl;
	// --------------------- Fill up the sparse matrix and compress it to remove extraneous elements ------//
  

        hipDeviceSynchronize();
	
        double JJ = 1.;

        cout<<"Running FillSparse"<<endl;

	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, vdim, d_H_vals, d_H_pos, d_Bond, lattice_Size, JJ);

        if( hipPeekAtLastError() != 0 ){
		cout<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}
		
	hipDeviceSynchronize(); //need to make sure all elements are initialized before I start compression

	status1 = hipFree(d_basis);
        status2 = hipFree(d_basis_Position);
        status3 = hipFree(d_Bond); // we don't need these later on
	
        if ( (status1 != hipSuccess) || 
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ){
          cout<<"Freeing bond and basis information failed! Error: "
          <<hipGetErrorString( hipPeekAtLastError() )<<endl;
          return 1;
        }

	bpg.x = vdim/32;
        bpg.y = ((2*lattice_Size)/32) + 1;
	tpb.x = 32;
        tpb.y = 32;

        cout<<"Running CompressSparse"<<endl;
       	
	CompressSparse<<<bpg, tpb>>>(d_H_vals, d_H_pos, vdim, lattice_Size);
        
        hipDeviceSynchronize();

        cout<<hipGetErrorString(hipPeekAtLastError() )<<endl;
        cout<<"Running SortHamiltonian"<<endl;

        
        SortHamiltonian<<<vdim/2, 64>>>(d_H_pos, d_H_vals, vdim, lattice_Size, 0);
        hipDeviceSynchronize();
        SortHamiltonian<<<vdim/2, 64>>>(d_H_pos, d_H_vals, vdim, lattice_Size, vdim/2);
        hipDeviceSynchronize();

        if (hipPeekAtLastError != 0){
          cout<<hipGetErrorString(hipPeekAtLastError())<<endl;
        }

	long2* buffer_H_pos;
	hipDoubleComplex* buffer_H_vals;

	hipHostAlloc(&buffer_H_pos, vdim*stridepos*sizeof(long2), hipHostMallocDefault);
	hipHostAlloc(&buffer_H_vals, vdim*strideval*sizeof(hipDoubleComplex), hipHostMallocDefault);

        hipDeviceSynchronize();

	hipMemcpy(buffer_H_pos, d_H_pos, vdim*stridepos*sizeof(long2), hipMemcpyDeviceToHost);
	hipMemcpy(buffer_H_vals, d_H_vals, vdim*strideval*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	hipMemcpy(h_H_pos, buffer_H_pos, vdim*stridepos*sizeof(long2), hipMemcpyHostToHost);
	hipMemcpy(h_H_vals, buffer_H_vals, vdim*strideval*sizeof(hipDoubleComplex), hipMemcpyHostToHost);

	hipHostFree(buffer_H_pos);
	hipHostFree(buffer_H_vals);

	for(int ii = 0; ii < vdim; ii++){
		num_Elem += (h_H_pos[ idx(ii, 0, stridepos) ]).y;
	}

        UpperHalfToFull(h_H_pos, h_H_vals, buffer_H_pos, buffer_H_vals, num_Elem, vdim, lattice_Size);
	
	hipHostFree(h_H_vals);
	hipHostFree(h_H_pos);

	dim3 tpb2 = ( tpb.x);	
	dim3 bpg2 = ( (2*num_Elem - vdim)/tpb.x );

	status1 = hipMalloc(&d_H_vals, (2*num_Elem - vdim)*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&d_H_pos, (2*num_Elem - vdim)*sizeof(long2));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
        	cout<<"Reallocating device Hamiltonian arrays failed! Error: "<< hipGetErrorString( hipPeekAtLastError() )<<endl;     
            	return 1;                                                                       
        }

        status1 = hipMemcpy(d_H_vals, buffer_H_vals, (2*num_Elem - vdim)*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        status2 = hipMemcpy(d_H_pos, buffer_H_pos, (2*num_Elem - vdim)*sizeof(long2), hipMemcpyHostToDevice);
                                                                                       
	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
            cout<<"Hamiltonian copy from host buffer to device failed!"<< hipGetErrorString( hipPeekAtLastError() )<<endl;     
            return 1;                                                                        
        }
        	
        hipHostFree(buffer_H_vals);
        hipHostFree(buffer_H_pos);
        
	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&hamil_PosRow, num_Elem*sizeof(long));
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(long));

	if ( (status1 != hipSuccess) ||
	     (status2 != hipSuccess) ||
	     (status3 != hipSuccess) ){
		cout<<"Memory allocation for COO representation failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}
        cout<<"Running FullToCOO."<<endl;
	
        FullToCOO<<<bpg2, tpb2>>>(num_Elem, d_H_vals, d_H_pos, hamil_Values, hamil_PosRow, hamil_PosCol, vdim); // csr and description initializations happen somewhere else


	hipFree(d_H_vals); //cleanup
	hipFree(d_H_pos);
       

	return 0;
}

int main(){
        //hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024); //have to set a heap size or malloc()s on the device will fail
        
	long* Bond;
	Bond = (long*)malloc(16*3*sizeof(long));
	
	Bond[0] = 0; 	Bond[1] = 1; 	Bond[2] = 2;	Bond[3] = 3;	Bond[4] = 4;	Bond[5] = 5;
	Bond[6] = 6; 	Bond[7] = 7;	Bond[8] = 8;	Bond[9] = 9;	Bond[10] = 10;	Bond[11] = 11;
	Bond[12] = 12;	Bond[13] = 13;	Bond[14] = 14;	Bond[15] = 15;	Bond[16] = 1; 	Bond[17] = 2;
	Bond[18] = 3; 	Bond[19] = 0;	Bond[20] = 5; 	Bond[21] = 6;	Bond[22] = 7;	Bond[23] = 4;
	Bond[24] = 9;	Bond[25] = 10; 	Bond[26] = 11;	Bond[27] = 8;	Bond[28] = 13;	Bond[29] = 14;
	Bond[30] = 15; 	Bond[31] = 12; 	Bond[32] = 4;	Bond[33] = 5;	Bond[34] = 6;	Bond[35] = 7;
	Bond[36] = 8;	Bond[37] = 9;	Bond[38] = 10;	Bond[39] = 11;	Bond[40] = 12;	Bond[41] = 13;
	Bond[42] = 14;	Bond[43] = 15;	Bond[44] = 0;	Bond[45] = 1;	Bond[46] = 2;	Bond[47] = 3;

	hipDoubleComplex* hamil_Values;

	long* hamil_PosRow;

	long* hamil_PosCol;


	int rtn = ConstructSparseMatrix(0, 16, Bond, hamil_Values, hamil_PosRow, hamil_PosCol);

	free(Bond);

	return rtn;
}
/* Function FillSparse: this function takes the empty Hamiltonian arrays and fills them up. Each thread in x handles one ket |i>, and each thread in y handles one site T0
Inputs: d_basis_Position - position information about the basis
	d_basis - other basis infos
	d_dim - the number of kets
	H_vals - an array that will store the values
	H_pos - an array that will store the positions of things
	d_Bond - the bond information
	d_lattice_Size - the number of lattice sites
	JJ - the coupling parameter 

*/

__global__ void FillSparse(long* d_basis_Position, long* d_basis, int dim, hipDoubleComplex* H_vals, long2* H_pos, long* d_Bond, int lattice_Size, const double JJ){

	int jj = blockIdx.y; //my indices!
	long ii = threadIdx.x + blockDim.x*blockIdx.x;
        int T0 = threadIdx.y;

        __shared__ long3 tempbond[16];
        __shared__ int count;
	count = 0;
        __shared__ long temppos[16];
        __shared__ hipDoubleComplex tempval[16]; //going to eliminate a huge number of read/writes to d_Bond, H_vals, H_pos in global memory

        if(jj < lattice_Size) {
        	(tempbond[jj]).x = d_Bond[jj];
		(tempbond[jj]).y = d_Bond[lattice_Size + jj];
		(tempbond[jj]).z = d_Bond[2*lattice_Size + jj];
        }
        

        __syncthreads();

        int stridepos = 2*lattice_Size + 2;
        int strideval = 2*lattice_Size + 1;

	int si, sj,sk,sl; //spin operators
	unsigned long tempi, tempj, tempod;
	hipDoubleComplex tempD;

        tempi = d_basis[ii];

	if( ii < dim ){
            if (T0 < lattice_Size){

		//Diagonal part----------------
                if (blockIdx.y == 0){
		    
		    temppos[0] = d_basis_Position[tempi];

		    tempval[0] = HDiagPart(tempi, lattice_Size, tempbond, JJ);

                    H_vals[ idx(ii, 0, strideval) ] = tempval[0];
                    (H_pos[ idx(ii, 1, stridepos) ]).y = temppos[0];
                    (H_pos[ idx(ii, 1, stridepos) ]).x = ii;
                }
                

		//-------------------------------
		//Horizontal bond ---------------
		si = (tempbond[T0]).x;
		tempod = tempi;
		sj = (tempbond[T0]).y;
	
		tempod ^= (1<<si);   //toggle bit 
		tempod ^= (1<<sj);   //toggle bit 

		if (d_basis_Position[tempod] > ii){ //build only upper half of matrix
        		temppos[T0] = d_basis_Position[tempod];
        		tempval[T0] = HOffBondX(T0,tempi, JJ);
			atomicAdd(&count,1); 
      		}

		else {
			temppos[T0] = -1;
			tempval[T0] = make_hipDoubleComplex(0., 0.);
		}

                __syncthreads();

                H_vals[ idx(ii, 2*T0, strideval) ] = tempval[T0];
                (H_pos[ idx(ii, 2*T0, stridepos) ]).y = temppos[T0];
                (H_pos[ idx(ii, 2*T0, stridepos) ]).x = ii;

		//Vertical bond -----------------
		tempod = tempi;
      		sj = (tempbond[T0]).z;

      		tempod ^= (1<<si);   //toggle bit 
     		tempod ^= (1<<sj);   //toggle bit
                 
      		if (d_basis_Position[tempod] > ii){ 
        		temppos[T0] = d_basis_Position[tempod];
   
        		tempval[T0] = HOffBondY(T0,tempi, JJ);
			atomicAdd(&count,1);
      		}

		else {
			temppos[T0] = -1;
			tempval[T0] = make_hipDoubleComplex(0., 0.);
		}

                //time to write back to global memory
                __syncthreads;
                (H_pos[ idx(ii, 0, stridepos) ]).x = ii;
                (H_pos[ idx(ii, 0, stridepos) ]).y, count;
                
                (H_pos[ idx(ii, 2*T0 + 1, stridepos) ]).x = ii;
                (H_pos[ idx(ii, 2*T0 + 1, stridepos) ]).y = temppos[T0];
                
                H_vals[ idx(ii, 2*T0 + 1, strideval) ] = tempval[T0];
                                 
            }//end of T0 

	}//end of ii
}//end of FillSparse

/* Function: CompressSparse - this function takes the sparse matrix with lots of "buffer" memory sitting on the end of each array, and compresses it down to get rid of the extra memory
Inputs:	H_vals - an array of arrays of Hamiltonian values
	H_pos - an array of arrays of value positions in columns
	d_dim - the dimension of the Hamiltonian, stored on the device
	lattice_Size - the number of lattice sites
Outputs: H_vals - an array of smaller arrays than before
	 H_pos - see above

*/
__global__ void CompressSparse(hipDoubleComplex* H_vals, long2* H_pos, long d_dim, const int lattice_Size){

	long row = blockDim.x*blockIdx.x + threadIdx.x;
        long col = blockDim.y*blockIdx.y + threadIdx.y;

	__shared__ int iter;
	iter = 0;

	if (row < d_dim){

		// the basic idea here is to have each x thread go to the ith row, and each y thread go to the jth element of that row. then using a set of __shared__ temp arrays, we read in the Hamiltonian values and do our comparisons n stuff on them


		const int size1 = 2*lattice_Size + 2;
		const int size2 = 2*lattice_Size + 1;
	
		__shared__ long s_H_pos[34]; //hardcoded for now because c++ sucks - should be able to change this later by putting all these functions in a separate .cu that doesn't use c++ functionality
		__shared__ hipDoubleComplex s_H_vals[33];
		

		if (col < size2){
			s_H_pos[col] = (H_pos[ idx( row, col, size1 ) ]).y;
			s_H_vals[col] = H_vals[ idx( row, col, size2) ];
                
                }

		if (col == size2){
			s_H_pos[col] = (H_pos[ idx(row, col, size1) ]).y;
		} //loading the Hamiltonian information into shared memory

                

		__syncthreads(); // have to make sure all loading is done before we start anything else


                

	        if (col < size2){
			if( (s_H_pos[col+1] != -1) ){
				
                                (H_pos[ idx(row, iter+1, size1) ]).y = s_H_pos[col+1];
                                H_vals[ idx(row, iter, size2) ] = s_H_vals[col];
                                //printf("%d %d %d \t", row, col, s_H_pos[col + 1]);
				atomicAdd(&iter,1);
			}
		 
		}

		
	}
}

//this function takes the upper half form I had from FillSparse and CompressSparse and fills out the bottom half of the matrix - since there are so many comparisons it's probably faster to just do this on CPU
__host__ void UpperHalfToFull(long2* H_pos, hipDoubleComplex* H_vals, long2* buffer_pos, hipDoubleComplex* buffer_val, long num_Elem, long dim, int lattice_Size) {

	int stridepos = 2*lattice_Size + 2;
	int strideval = 2*lattice_Size + 1;

	hipHostAlloc(&buffer_pos, (2*num_Elem - dim)*sizeof(long2), hipHostMallocDefault);
	hipHostAlloc(&buffer_val, (2*num_Elem - dim)*sizeof(hipDoubleComplex), hipHostMallocDefault);

	unsigned long start = 0;

	for(long ii = 0; ii<dim; ii++){
                //cout<<start<<endl;         
		long size = dim - (H_pos[ idx(ii, 0, stridepos) ]).y ; // the maximum number of nonzero elements we could pull from the columns
		hipDoubleComplex* temp;
		long* temp_col;

		temp = (hipDoubleComplex*)malloc(size*sizeof(hipDoubleComplex));
		temp_col = (long*)malloc(size*sizeof(long));

		long iter = 0;

		for(int jj = 0; jj<ii; jj++){
			for(int kk = 1; kk <= (H_pos[ idx(jj, 0, stridepos) ]).y; kk++){

				if((H_pos[ idx(jj, kk, stridepos) ]).y = ii){
					cout<<iter<<endl;
					temp[iter] = H_vals[ idx(jj, kk-1, strideval) ];
					temp_col[iter] = jj;
					iter++;
				}

			}
		}


		hipDoubleComplex temp_vals[(H_pos[ idx(ii, 0, stridepos) ]).y + iter];
		long temp_pos[(H_pos[ idx(ii, 0, stridepos) ]).y + iter + 1];
		temp_pos[0] = iter + (H_pos[ idx(ii , 0, stridepos) ] ).y; //we'll need this number later!
		//cout<<iter<<" "<<(H_pos[ idx(ii, 0, stridepos) ].y)<<endl;
		for(int ll = 0; ll < (H_pos[ idx(ii, 0, stridepos) ]).y + iter; ll++){
			if (ll < iter){
				temp_vals[ll] = temp[ll];
				temp_pos[ll+1] = temp_col[ll];

			}
			else{
				temp_vals[ll] = H_vals[ idx(ii, ll - iter, strideval) ];
				temp_pos[ll+1] = (H_pos[ idx(ii, ll - iter + 1, stridepos) ]).y;
			}

                        
			buffer_val[ start + ll ] = temp_vals[ll];
			(buffer_pos[ start + ll ]).x = ii;
			(buffer_pos[ start + ll ]).y = temp_pos[ll + 1];
		}
				
		start += temp_pos[0];
	}

} 

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
	 H_vals - the Hamiltonian values
	 H_pos - the Hamiltonian positions
	 hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(long num_Elem, hipDoubleComplex* H_vals, long2* H_pos, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol, long dim){

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.x;

	long size = 2*num_Elem - dim;

	long start = 0;

	__shared__ long2 s_H_pos[256]; //hardcoded for now because c++ sucks
	__shared__ hipDoubleComplex s_H_vals[256];

	if (i < size){
		(s_H_pos[j]).x = (H_pos[i]).x;
		(s_H_pos[j]).y = (H_pos[i]).y;
		s_H_vals[j] = H_vals[i];
	
		__syncthreads();
		
		hamil_Values[i] = s_H_vals[j];
		hamil_PosRow[i] = (s_H_pos[j]).x;
		hamil_PosCol[i] = (s_H_pos[j]).y;
		
	}
}

__global__ void SortHamiltonian( long2* H_pos, hipDoubleComplex* H_vals, long dim, int lattice_Size, long start){
	long i = blockIdx.x + start;
	int j = threadIdx.x;

	__shared__ int maxpos;
	maxpos = 0;
	__shared__ int maxnumdigits;
	__shared__ int rowlength;

	__shared__ hamstruct temparray[33];
	__shared__ int sortarray[10];

	if ( i < dim){

      		rowlength = (H_pos[ idx(i, 0, 2*lattice_Size + 2) ]).y;
      	
		if (j < rowlength ){
      			(temparray[j]).position  = ( H_pos[ idx(i, j + 1, 2*lattice_Size + 2) ] ).y;
              		(temparray[j]).value = H_vals[ idx(i, j, 2*lattice_Size + 1) ];
                        //printf("%d \n", (temparray[j]).position);
              		sortarray[j] = 0;

              		atomicMax(&maxpos, (temparray[j]).position ); //computes maximum using bithax
        	       	__syncthreads(); //need to finish loading everything            
	       		maxnumdigits = floor ( log10f( maxpos ) ) + 1; 

			maxpos = 0; //maxpos now functions as total
	
			__syncthreads();

			for(int k = 1; k < maxnumdigits; k++){
				int key = nthdigit(temparray[j].position, 1);
				sortarray[key] = sortarray[key] + 1;
				
  			
				if ( j < 10 ){				
					int temp = sortarray[j];
					sortarray[j] = maxpos;
					atomicAdd(&maxpos, temp);
				}

				temparray[sortarray[key]] = temparray[j];
				sortarray[key] = sortarray[key] + 1;

                                __syncthreads();

			}//k

			(H_pos[ idx(i, j, 2*lattice_Size + 2) ]).y = temparray[j].position;
			H_vals[ idx(i, j, 2*lattice_Size + 1) ] = temparray[j].value;
			//passing the sorted array back to global memory
					
		}//j

	}//i
}


