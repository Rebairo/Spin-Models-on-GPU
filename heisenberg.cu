//#include"hamiltonian.h"
#include"lattice.h"
#include<cstdlib>
#include"hip/hip_runtime.h"
#include<iostream>
#include"lanczos.h"
//#include"hamiltonian.h"
int main()
{
    for(int i = 0; i < 1; i++){
    int** Bond;
    //cout<<i<<" "<<endl;
    int how_many = 1;
    /*if (i == 1)
    {
        how_many = 5;
    }*/
    Bond = (int**)malloc(how_many*sizeof(int*));
    d_hamiltonian* hamil_lancz = (d_hamiltonian*)malloc(how_many*sizeof(d_hamiltonian));
    int* nsite = (int*)malloc(how_many*sizeof(int));
    int* Sz = (int*)malloc(how_many*sizeof(int));
    float* JJ = (float*)malloc(how_many*sizeof(float));
    int* model_type = (int*)malloc(how_many*sizeof(int));
    int* num_Elem = (int*)malloc(how_many*sizeof(int));

    //hipSetDevice(1);
    int device = 0; //i%2;

    for(int i = 0; i < how_many; i++)
    {
        
        nsite[i] = 12;
        Bond[i] = (int*)malloc(3*nsite[i]*sizeof(int));
        Fill_Bonds_12A(Bond[i]);
        Sz[i] = 0;
        JJ[i] = 1.f;
        model_type[i] = 0;
    }


    int dim;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

    hipEventRecord(start,0);
    ConstructSparseMatrix(how_many, model_type, nsite, Bond, hamil_lancz, JJ, Sz, num_Elem, device);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout<<"Time to construct Hamiltonians: "<<time<<endl;
    hipEventRecord(start,0);
    lanczos(how_many, num_Elem, hamil_lancz, 200, 3, 1e-12);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout<<"Time to perform Lanczos: "<<time<<endl;
    for(int j = 0; j<how_many; j++)
    {
        hipFree(hamil_lancz[j].rows);
        hipFree(hamil_lancz[j].cols);
        hipFree(hamil_lancz[j].vals);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(nsite);
    free(Sz);
    free(JJ);
    free(model_type);
    free(Bond);
    free(hamil_lancz);
    free(num_Elem);
    }
    return 0;
}
