#include "hip/hip_runtime.h"
// Katharine Hyatt
// A set of functions to implement the Lanczos method for a generic Hamiltonian
// Based on the codes Lanczos_07.cpp and Lanczos07.h by Roger Melko
//-------------------------------------------------------------------------------

#include"lanczos.h"

// h_ means this variable is going to be on the host (CPU)
// d_ means this variable is going to be on the device (GPU)
// s_ means this variable is shared between threads on the GPU
// The notation <<x,y>> before a function defined as global tells the GPU how many threads per block to use
// and how many blocks per grid to use
// blah.x means the real part of blah, if blah is a data type from hip/hip_complex.h
// blah.y means the imaginary party of blah, if blah is a data type from hip/hip_complex.h
// threadIdx.x (or block) means the "x"th thread from the left in the block (or grid)
// threadIdx.y (or block) means the "y"th thread from the top in the block (or grid)

//Function vecdiff: calculates the difference between some vectors, two of which is multiplied by a scalar
//Implements w = x - a*y - b*z 
//-------------------------------------------------------------------------------------------------------
//Input: w, a "dummy pointer" to the vector that is changed
//       x, the vector that a*y and b*z are subtracted from 
//       alpha, the scalar the first subtracted vector is multiplied by
//       y, the first subtracted vector
//       beta, the scalar the second subtraced vector is multiplied by
//       z, the second subtracted vector
//       n, the number of elements in all the vectors
//       Note: no control is put in here to make sure all vectors are the same size. 
//------------------------------------------------------------------------------------------------------
//Output: w, the result of the subtractions
//        All other quantities remain unchanged
//------------------------------------------------------------------------------------------------------
__global__ void vecdiff(hipDoubleComplex* w, hipDoubleComplex* x, hipDoubleComplex alpha, hipDoubleComplex* y, hipDoubleComplex beta, hipDoubleComplex* z, int n){

  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n) {
    w[i] = hipCsub(x[i],hipCsub(hipCmul(alpha,y[i]),hipCmul(beta,z[i]))); //this is the dirtiest thing ever
  }
  __syncthreads();
}

//Function assignr: assigns the real parts of an array of double complex numbers the value of some double
//-------------------------------------------------------------------------------------------------------
//Input: a, a vector of double precision complex numbers whose real parts we would like to change
//       b, the real number that will become the real part of the complex numbers in a
//       n, the number of elements in a
//-------------------------------------------------------------------------------------------------------
//Output: a, the vector of complex numbers whose real parts have been changed
//        All other quantities are unchanged
//------------------------------------------------------------------------------------------------------- 
__global__ void assignr(hipDoubleComplex* a, double b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n){
    a[i] = make_hipDoubleComplex(b,0.);
  }
}

__global__ void assign(double* a, double b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if (i < n){
    a[i] = b;
  }
}


//Function complextodoubler: assigns the real parts of complex numbers in an array to doubles in another array
//------------------------------------------------------------------------------------------------------------
//Input: a, the vector of complex numbers whose real parts we are extracting
//       b, the vector of doubles that will hold the real parts
//       n, the number of elements in the vectors
//------------------------------------------------------------------------------------------------------------
//Output: b, the vector of doubles now holding the real parts
//        All other quantities are unchanged
//------------------------------------------------------------------------------------------------------------
__global__ void complextodoubler(hipDoubleComplex* a, double* b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if(i <= n){
    b[i] = a[i].x; 
  }
}

//Same as above, but in this case the parts are shifted by one space in the vector
__global__ void complextodoubler2(hipDoubleComplex* a, double* b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x + 1;

  if(i <= n){
    b[i-1] = a[i].x;
  }
  if(i == n+1){
    b[i-1] = 0.;
  }
} 

__global__ void zero(double** a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int j = blockDim.y*blockIdx.y + threadIdx.y;

  if ( i< m && j < m){
    a[i][j] = 0. ;
  }
}
// Note: to get the identity matrix, apply the fuction zero above first
__global__ void identity(double** a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (i < m ){

    a[i][i] = 1.;
  }
}

__global__ void arraysalloc(hipDoubleComplex** a, int n, int m){
  int i = threadIdx.x + m;
  a[i] = (hipDoubleComplex*)malloc(n*sizeof(hipDoubleComplex));
}

//Function lanczos: takes a hermitian matrix H, tridiagonalizes it, and finds the n smallest eigenvalues - this version only returns eigenvalues, not
// eigenvectors. Doesn't use sparse matrices yet either, derp. Should be a very simple change to make using CUSPARSE, which has functions for operations
// between sparse matrices and dense vectors
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Input: h_H, a Hermitian matrix of complex numbers (not yet sparse)
//        dim, the dimension of the matrix
//        max_Iter, the starting number of iterations we'll try
//        num_Eig, the number of eigenvalues we're interested in seeing
//        conv_req, the convergence we'd like to see
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Output: h_ordered, the array of the num_Eig smallest eigenvalues, ordered from smallest to largest
//---------------------------------------------------------------------------------------------------------------------------------------------------        

int main(){}

void lanczos(const hipDoubleComplex* h_H, const int dim, int max_Iter, const int num_Eig, const double conv_req){

  hipblasStatus_t linalgstat;
  linalgstat = cublasInit(); //have to initialize the cuBLAS environment, or my program won't work! I could use this later to check for errors as well

  hipsparseHandle_t sparsehandle;
  hipsparseStatus_t sparsestatus = hipsparseCreate(&sparsehandle); //have to initialize the cusparse environment too! This variable gets passed to all my cusparse functions

  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    printf("Failed to initialize CUBLAS! \n");
  }

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    printf("Failed to initialize CUSPARSE! \n");
  }

  hipError_t status1, status2, status3, status4; //this is to throw errors in case things (mostly memory) in the code fail!  

  size_t h_pitch;
  size_t d_pitch = h_pitch;

  hipDoubleComplex* d_H;

  status1 = hipMallocPitch(&d_H, &h_pitch, dim*sizeof(hipDoubleComplex), dim); //allocating memory in the GPU for our matrix
  if (status1 != hipSuccess){
    printf("Hamiltonian device memory allocation failed! \n");
  }

  status2 = hipMemcpy2D(d_H, d_pitch, &h_H, h_pitch, dim, dim, hipMemcpyHostToDevice ); //copying the matrix into the GPU
  // the above memory code could be total lunacy

  if (status2 != hipSuccess){
    printf("Hamiltonian copy from host to device failed! \n");
  }

  //Now that I have the Hamiltonian on the GPU, it's time to start generating eigenvectors

  hipDoubleComplex* d_a; //these are going to store the elements of the tridiagonal matrix
  hipDoubleComplex* d_b; //they have to be hipDoubleComplex because that's the only input type the cublas functions I need will take

  status3 = hipMalloc(&d_a, max_Iter*sizeof(hipDoubleComplex));
  status4 = hipMalloc(&d_b, max_Iter*sizeof(hipDoubleComplex));

  if ((status3 != hipSuccess) || (status4 != hipSuccess)){
    printf("Matrix elements memory allocation failed! \n");
  }


  int tpb = 256; //threads per block - a conventional number
  int bpg = (dim + tpb - 1)/tpb; //blocks per grid

  //Making the "random" starting vector

  hipDoubleComplex** d_eigen_Array; //this thing is an array of pointers to the eigenvectors 
  status1 = hipMalloc(&d_eigen_Array, max_Iter*sizeof(hipDoubleComplex*)); // making the pointer array

  if (status1 != hipSuccess){
    printf("Eigenvector array allocation failed! \n");
  }

  arraysalloc<<<1, max_Iter>>>(d_eigen_Array, dim, 0); //time to make the actual arrays of the eigenvectors

  assignr<<<bpg,tpb>>>(d_eigen_Array[0], 1., dim); //assigning the values of the "random" starting vector
  
  hipDoubleComplex alpha = make_hipDoubleComplex(1.,0.);
  hipDoubleComplex beta = make_hipDoubleComplex(0.,0.); 

  hipblasZgemv('N', dim, dim, alpha, d_H, dim*sizeof(hipDoubleComplex), d_eigen_Array[0], sizeof(hipDoubleComplex), beta, d_eigen_Array[1], sizeof(hipDoubleComplex)); // the Hamiltonian is applied here

  //*********************************************************************************************************
  // This is just the first steps so I can do the rest  
  d_a[0] = hipblasZdotc(dim, d_eigen_Array[0], sizeof(hipDoubleComplex), d_eigen_Array[1], sizeof(hipDoubleComplex));
  d_b[0] = make_hipDoubleComplex(0.,0.);

  hipDoubleComplex* y;
  status2 = hipMalloc(&y, dim*sizeof(hipDoubleComplex));

  if (status2 != hipSuccess){
    printf("Memory allocation of y dummy vector failed! \n");
  }
  
  assignr<<<bpg,tpb>>>(y,0., dim); //a dummy vector of 0s that i can stick in my functions

  vecdiff<<<bpg,tpb>>>(d_eigen_Array[1], d_eigen_Array[1], d_a[0], d_eigen_Array[0], y[0], y, dim);
  d_b[1] = make_hipDoubleComplex(sqrt(hipblasDznrm2(dim, d_eigen_Array[1], sizeof(hipDoubleComplex))),0.);
  // this function (above) takes the norm
  
  hipDoubleComplex gamma = make_hipDoubleComplex(1./d_b[1].x,0.); //alpha = 1/beta in v1 = v1 - alpha*v0

  hipblasZaxpy(dim, gamma, d_eigen_Array[1], sizeof(hipDoubleComplex), y, sizeof(hipDoubleComplex)); // function performs a*x + y

  //Now we're done the first round!
  //*********************************************************************************************************

  double* d_ordered;
  status1 = hipMalloc(&d_ordered, num_Eig*sizeof(double));

  if (status1 != hipSuccess){
    printf("Eigenvalue array memory allocation failed! \n");
  }

  assign<<<bpg,tpb>>>(d_ordered, 0., num_Eig);

  double* gs_Energy;
  *gs_Energy = 1.; //the lowest energy

  int returned;

  int iter = 0;

  // In the original code, we started diagonalizing from iter = 5 and above. I start from iter = 1 to minimize issues of control flow
  double* d_diag;
  double* d_offdia;

  status3 = hipMalloc(&d_diag, num_Eig*sizeof(double));
  status4 = hipMalloc(&d_offdia, num_Eig*sizeof(double));

  if ((status3 != hipSuccess) || (status4 != hipSuccess)){
    printf("Second matrix elements array memory allocation failed! \n");
  }

  double* eigtemp;
  *eigtemp = 0.;

  while( abs(*gs_Energy - *eigtemp)> conv_req){ //this is a cleaner version than what was in the original - way fewer if statements

    iter++;

    status1 = hipMemcpy(eigtemp, &d_ordered[num_Eig - 1], sizeof(double), hipMemcpyDeviceToHost);

    if (status1 != hipSuccess){
      printf("Copying last eigenvalue failed \n");
    }

    hipblasZgemv('N', dim, dim, alpha, d_H, dim*sizeof(hipDoubleComplex), d_eigen_Array[iter], sizeof(hipDoubleComplex), beta, d_eigen_Array[iter+1], sizeof(hipDoubleComplex)); // the Hamiltonian is applied here, in this gross expression

    d_a[iter] = hipblasZdotc(dim, d_eigen_Array[iter], sizeof(hipDoubleComplex), d_eigen_Array[iter + 1], sizeof(hipDoubleComplex));

    vecdiff<<<bpg,tpb>>>(d_eigen_Array[iter+1], d_eigen_Array[iter+1], d_a[iter], d_eigen_Array[iter], d_b[iter], d_eigen_Array[iter - 1], dim);

    d_b[iter+1] = make_hipDoubleComplex(sqrt(hipblasDznrm2(dim, d_eigen_Array[iter+1], sizeof(hipDoubleComplex))),0.);
    
    gamma = make_hipDoubleComplex(1./d_b[iter+1].x,0.);
    hipblasZaxpy(dim, gamma, d_eigen_Array[iter+1], sizeof(hipDoubleComplex), y, sizeof(hipDoubleComplex));
    
    //hipblasZcopy(dim, d_v_Mid, sizeof(hipDoubleComplex), d_v_Start, sizeof(hipDoubleComplex)); //switching my vectors around for the next iteration
    //hipblasZcopy(dim, d_v_End, sizeof(hipDoubleComplex), d_v_Mid, sizeof(hipDoubleComplex)); unnecesarry now that i'm using the array of pointers

    d_diag[iter] = 0.; //adding another spot in the tridiagonal matrix representation
    d_offdia[iter] = 0.;

    complextodoubler<<<bpg,tpb>>>(d_a, d_diag, iter);
    complextodoubler2<<<bpg,tpb>>>(d_b, d_offdia, iter);

    double** d_H_eigen;
    size_t d_eig_pitch;

    status1 = hipMallocPitch(&d_H_eigen, &d_eig_pitch, iter*sizeof(double), iter);
    if (status1 != hipSuccess){
      printf("tqli eigenvectors matrix memory allocation failed! \n");
    }
    
    zero<<<bpg,tpb>>>(d_H_eigen, iter);
    identity<<<bpg,tpb>>>(d_H_eigen, iter); //set this matrix to the identity

    returned = tqli(d_diag, d_offdia, iter + 1, d_H_eigen); //tqli is in a separate file   

    assign<<<tpb,bpg>>>(d_ordered, d_diag[0], num_Eig);
    
    for(int i = 1; i < sizeof(d_eigen_Array); i++){ //todo: rewrite this as a setup where if you want 
      for(int j = 0; j< num_Eig; j++){// n smallest eigenvalues, you take the first n
        if (d_diag[i]< d_ordered[j]){ //elements, sort them, then add one element at a time 
          d_ordered[j] = d_diag[i]; // and binary search to see if it is smaller than any other
          break;
        }
      }
    } // or just use radixsort!

    status2 = hipMemcpy(&gs_Energy, &(d_ordered[num_Eig - 1]), sizeof(double), hipMemcpyDeviceToHost);

    if (status2 != hipSuccess){
      printf("Copying the eigenvalue failed! \n");
    }

    if (iter == sizeof(d_eigen_Array) - 2){// have to use this or d_b will overflow
      //this stuff here is used to resize the main arrays in the case that we aren't converging quickly enough
      //------------------------------------------------------------------------
	hipDoubleComplex* temp;
        status1 = hipMalloc(&temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        
        status2 = hipMemcpy(temp, d_a, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
        
        hipFree(d_a);
        status3 = hipMalloc(&d_a, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        status4 = hipMemcpy(d_a, temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
        
        if( (status1 != hipSuccess) ||
            (status2 != hipSuccess) ||
            (status3 != hipSuccess) ||
            (status4 != hipSuccess) ){
          printf("Resizing d_a failed! \n");
        } 

        status1 = hipMemcpy(temp, d_b, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

        hipFree(d_b);
        status2 = hipMalloc(&d_b, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        status3 = hipMemcpy(d_b, temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

        if ( (status1 != hipSuccess) ||
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ){
          printf("Resizing d_b failed! \n");
        }
          
        hipFree(temp);
        hipMalloc(&temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex*));
        
        status1 = hipMemcpy(temp, d_eigen_Array, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex*), hipMemcpyDeviceToDevice);

        int temp2 = sizeof(d_eigen_Array);

        status2 = hipFree(d_eigen_Array);
        status3 = hipMalloc(&d_eigen_Array, (2*temp2 + 1)*sizeof(hipDoubleComplex*));
        status4 = hipMemcpy(d_eigen_Array, temp, (2*temp2 + 1)*sizeof(hipDoubleComplex*), hipMemcpyDeviceToDevice);

        if ( (status1 != hipSuccess) ||
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ||
             (status4 != hipSuccess) ){
          printf("Resizing d_eigen_Array failed! \n");
        }

        arraysalloc<<<1, temp2 + 1>>>(d_eigen_Array, dim, temp2);
                
        hipFree(temp); 
        //resizing

    }   
  } 

  double* h_ordered;

  status1 = hipHostMalloc(&h_ordered, num_Eig*sizeof(double)); //a place to put the eigenvalues on the CPU

  if (status1 != hipSuccess){
    printf("Memory allocation for host eigenvector array failed! \n");
  }

  status2 = hipMemcpy(h_ordered, d_ordered, num_Eig*sizeof(double), hipMemcpyDeviceToHost); // moving the eigenvalues over

  if (status2 != hipSuccess){
    printf("Copying eigenvalues from GPU to CPU failed! \n");
  }

  for(int i = 0; i < num_Eig; i++){
    printf("%d \n", h_ordered[i]);
  } //write out the eigenenergies

  hipFree(&alpha);
  hipFree(&beta);
  hipFree(d_a);
  hipFree(d_b); //dropping stuff off
  // call the expectation values function
  
  // time to copy back all the eigenvectors
  //int* sizeptr;
  //hipMemcpy(sizeptr, &sizeof(d_eigen_Array), sizeof(int), hipMemcpyDeviceToHost);
  
  max_Iter = sizeof(d_eigen_Array);

  hipDoubleComplex** h_eigen_Array;
  status1=hipHostMalloc(&h_eigen_Array, max_Iter*sizeof(hipDoubleComplex*));

  if (status1 != hipSuccess){
    printf("CPU eigenvector array memory allocation failed! \n");
  }
  
  for(int i = 0; i < max_Iter; i++){
     status2 = hipHostMalloc(&h_eigen_Array[i], dim*sizeof(hipDoubleComplex*));
     status3 = hipMemcpy(h_eigen_Array[i], d_eigen_Array[i], dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
     if ( (status2 != hipSuccess) || (status3 != hipSuccess)){
       printf("Copying an eigenvector array failed! \n");
     }

  } // now the eigenvectors are available on the host CPU

  linalgstat = cublasShutdown();
	
  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    printf("CUBLAS failed to shut down properly! \n");
  }

  sparsestatus = hipsparseDestroy(sparsehandle);

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    printf("CUSPARSE failed to release handle! \n");
  }
}
// things left to do:
// write a thing (separate file) to call routines to find expectation values, should be faster on GPU 
// make the tqli thing better!
// do the hamiltonian generating stuff

//Function Hoperate: applies H to some vector to give a = H*b
//NOTE: this function CANNOT be called from the CPU.
//Only variables and functions on the GPU may access it.
//------------------------------------------------------------
//Input: H, a matrix of complex numbers
//       v0, the vector H is applied to
//       v1, a pointer to  the output vector
//       alpha, a dummy variable
//       beta, a dummy variable
//       dim, the number of elements in the vectors and the length of one side of H
//------------------------------------------------------------
//Output: v1, the result of H*v0
//        All other quantities are unchanged
//------------------------------------------------------------
//__device__ void Hoperate(hipDoubleComplex* H, hipDoubleComplex* v0, hipDoubleComplex v1, hipDoubleComplex alpha, hipDoubleComplex beta, int dim){
  
//  v1 = hipblasZgemv(N, dim, dim, alpha, &H, m*sizeof(hipDoubleComplex), v0, sizeof(hipDoubleComplex), beta, v1, sizeof(hipDoubleComplex));

//}

//fin
