#include "hip/hip_runtime.h"
// Katharine Hyatt
// A set of functions to implement the Lanczos method for a generic Hamiltonian
// Based on the codes Lanczos_07.cpp and Lanczos07.h by Roger Melko
//-------------------------------------------------------------------------------

#include"lanczos.h"

// h_ means this variable is going to be on the host (CPU)
// d_ means this variable is going to be on the device (GPU)
// s_ means this variable is shared between threads on the GPU
// The notation <<x,y>> before a function defined as global tells the GPU how many threads per block to use
// and how many blocks per grid to use
// blah.x means the real part of blah, if blah is a data type from hip/hip_complex.h
// blah.y means the imaginary party of blah, if blah is a data type from hip/hip_complex.h
// threadIdx.x (or block) means the "x"th thread from the left in the block (or grid)
// threadIdx.y (or block) means the "y"th thread from the top in the block (or grid)


__global__ void zero(hipDoubleComplex* a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if ( i < m){
    a[i] = make_hipDoubleComplex(0., 0.) ;
  }
}


__global__ void zero(double* a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if ( i < m ){
    a[i] = 0.;
  }
}

// Note: to get the identity matrix, apply the fuction zero above first
__global__ void unitdiag(double* a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (i < m ){

    a[i + m*i] = 1.;
  }
}


//Function lanczos: takes a hermitian matrix H, tridiagonalizes it, and finds the n smallest eigenvalues - this version only returns eigenvalues, not
// eigenvectors. Doesn't use sparse matrices yet either, derp. Should be a very simple change to make using CUSPARSE, which has functions for operations
// between sparse matrices and dense vectors
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Input: h_H, a Hermitian matrix of complex numbers (not yet sparse)
//        dim, the dimension of the matrix
//        max_Iter, the starting number of iterations we'll try
//        num_Eig, the number of eigenvalues we're interested in seeing
//        conv_req, the convergence we'd like to see
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Output: h_ordered, the array of the num_Eig smallest eigenvalues, ordered from smallest to largest
//---------------------------------------------------------------------------------------------------------------------------------------------------        

__host__ void lanczos(const int num_Elem, hipDoubleComplex*& d_H_vals, int*& d_H_rows, int*& d_H_cols, const int dim, int max_Iter, const int num_Eig, const double conv_req){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;

  hipblasStatus_t linalgstat;
  //have to initialize the cuBLAS environment, or my program won't work! I could use this later to check for errors as well
  hipblasHandle_t linalghandle;
	hipEventRecord(start, 0);  
	linalgstat = hipblasCreate(&linalghandle);

  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    std::cout<<"Initializing CUBLAS failed! Error: "<<linalgstat<<std::endl;
  }

  hipsparseHandle_t sparsehandle;
  hipsparseStatus_t sparsestatus = hipsparseCreate(&sparsehandle); //have to initialize the cusparse environment too! This variable gets passed to all my cusparse functions

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    std::cout<<"Failed to initialize CUSPARSE! Error: "<<sparsestatus<<std::endl;
  }
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Time to initialize libraries: "<<time<<std::endl;

	hipEventRecord(start,0);
  hipsparseMatDescr_t H_descr = 0;
  sparsestatus = hipsparseCreateMatDescr(&H_descr);
  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    std::cout<<"Error creating matrix description: "<<sparsestatus<<std::endl;
  }
  sparsestatus = hipsparseSetMatType(H_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    std::cout<<"Error setting matrix type: "<<sparsestatus<<std::endl;
  }
  sparsestatus = hipsparseSetMatIndexBase(H_descr, HIPSPARSE_INDEX_BASE_ZERO);
  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    std::cout<<"Error setting matrix index base: "<<sparsestatus<<std::endl;
  }
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Runtime to create description: "<<time<<std::endl;

  hipError_t status1, status2, status3, status4;

	int* d_H_rowptrs;
	status1 = hipMalloc(&d_H_rowptrs, (dim + 1)*sizeof(int));
	if (status1 != hipSuccess){ 
		std::cout<<"Error allocating d_H_rowptrs: "<<hipGetErrorString(status1)<<std::endl;
	}

	hipEventRecord(start, 0);
	hipsparseHybMat_t hyb_Ham;	
	hipsparseCreateHybMat(&hyb_Ham);

	sparsestatus = hipsparseXcoo2csr(sparsehandle, d_H_rows, num_Elem, dim, d_H_rowptrs, HIPSPARSE_INDEX_BASE_ZERO);
	sparsestatus = hipsparseZcsr2hyb(sparsehandle, dim, dim, H_descr, d_H_vals, d_H_rowptrs, d_H_cols, hyb_Ham, 0, HIPSPARSE_HYB_PARTITION_AUTO);
	 
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Failed to switch from COO to CSR! CUSPARSE Error: "<<sparsestatus<<std::endl;
	}

	if (hipPeekAtLastError() != hipSuccess){
		std::cout<<"Failed to switch from COO to CSR! Error: "<<hipGetErrorString( hipPeekAtLastError())<<std::endl;
	}

	//std::cout<<"Runtime to convert to CSR: "<<time<<std::endl;

	thrust::host_vector<hipDoubleComplex> h_a(max_Iter);

	thrust::host_vector<hipDoubleComplex> h_b(max_Iter);

	//hipDoubleComplex* d_a_ptr;
	//hipDoubleComplex* d_b_ptr; //we need these to pass to kernel functions 

	//Making the "random" starting vector

	thrust::device_vector<hipDoubleComplex> d_lanczvec(dim*max_Iter); //this thing is an array of the Lanczos vectors 
 
	hipDoubleComplex* lancz_ptr = thrust::raw_pointer_cast(&d_lanczvec[0]);

	hipDoubleComplex* v0;
	hipDoubleComplex* v1;
	hipDoubleComplex* v2;
	status1 = hipMalloc(&v0, dim*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&v1, dim*sizeof(hipDoubleComplex));
	status3 = hipMalloc(&v2, dim*sizeof(hipDoubleComplex));

  //thrust::device_ptr<hipDoubleComplex> v0_ptr(v0);
  //thrust::device_ptr<hipDoubleComplex> v1_ptr(v1);
  //thrust::device_ptr<hipDoubleComplex> v2_ptr(v2);

  //thrust::device_vector<hipDoubleComplex> v0(dim);
  //thrust::device_vector<hipDoubleComplex> v1(dim);
  //thrust::device_vector<hipDoubleComplex> v2(dim);
  //hipDoubleComplex* v0_ptr = thrust::raw_pointer_cast(&v0[0]);
  //hipDoubleComplex* v1_ptr = thrust::raw_pointer_cast(&v1[0]);
  //hipDoubleComplex* v2_ptr = thrust::raw_pointer_cast(&v2[0]);

	hipEventRecord(start, 0);
	hipDoubleComplex* host_v0 = (hipDoubleComplex*)malloc(dim*sizeof(hipDoubleComplex));
  for(int i = 0; i<dim; i++){
    host_v0[i] = make_hipDoubleComplex(0. , 0.);
    if (i%4 == 0) host_v0[i] = make_hipDoubleComplex(1.0, 0.) ;
    else if (i%5 == 0) host_v0[i] = make_hipDoubleComplex(-2.0, 0.);
    else if (i%7 == 0) host_v0[i] = make_hipDoubleComplex(3.0, 0.);
    else if (i%9 == 0) host_v0[i] = make_hipDoubleComplex(-4.0, 0.);

  }

	hipMemcpy(v0, host_v0, dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Time to set and push v0: "<<time<<std::endl;

	double normtemp;

	hipEventRecord(start,0);
	linalgstat = hipblasDznrm2(linalghandle, dim, v0, 1, &normtemp);
	normalize<<<dim/512 + 1, 512>>>(v0, dim, normtemp);

	hipDoubleComplex alpha = make_hipDoubleComplex(1.,0.);
	hipDoubleComplex beta = make_hipDoubleComplex(0.,0.); 

	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Time to normalize v0: "<<time<<std::endl;

	hipEventRecord(start,0);
	sparsestatus = hipsparseZhybmv(sparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, H_descr, hyb_Ham, v0, &beta, v1); // the Hamiltonian is applied here

	if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Getting V1 = H*V0 failed! Error: ";
		std::cout<<sparsestatus<<std::endl;
	}
	hipDeviceSynchronize();
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Time to get V1=H*V0: "<<time<<std::endl;
	
	if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Getting V1 = H*V0 failed! Error: ";
		std::cout<<sparsestatus<<std::endl;
	}
	if (hipPeekAtLastError() != 0 ){
		std::cout<<"Getting V1  = H*V0 failed! Error: ";
		std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
	} 

  //*********************************************************************************************************
  
  // This is just the first steps so I can do the rest
  
  /*try{ 
    d_a_ptr = raw_pointer_cast(&d_a[0]);  
  }
  catch( thrust::system_error e ){
    std::cerr<<"Error settng d_a_ptr: "<<e.what()<<std::endl;
    exit(-1);
  }*/

	hipDoubleComplex dottemp = make_hipDoubleComplex(0. ,0.);
   
	linalgstat = hipblasZdotc(linalghandle, dim, v1, 1, v0, 1, &dottemp); 
	if (linalgstat != HIPBLAS_STATUS_SUCCESS){
		std::cout<<"Getting d_a[0] failed! Error: ";
		std::cout<<linalgstat<<std::endl;
	  }

	h_a[0] = dottemp;
	//hipMemcpy(d_a_ptr, &dottemp, sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	if (linalgstat != HIPBLAS_STATUS_SUCCESS){
		std::cout<<"Getting h_a[0] failed! Error: ";
		std::cout<<linalgstat<<std::endl;
	}

	h_b[0] = make_hipDoubleComplex(0., 0.);

	hipDoubleComplex* y;
	status2 = hipMalloc(&y, dim*sizeof(hipDoubleComplex));

	if (status2 != hipSuccess){
		std::cout<<"Memory allocation of y dummy vector failed! Error:";
		std::cout<<hipGetErrorString( status2 )<<std::endl;
	}
  
	zero<<<dim/512 + 1, 512>>>(y, dim);
	hipDeviceSynchronize();
  
	hipDoubleComplex axpytemp = hipCmul(make_hipDoubleComplex(-1.,0), h_a[0]);

	linalgstat = hipblasZaxpy(linalghandle, 0, &axpytemp, v0, 1, v1, 1);
	//std::cout<<axpytemp.x<<" "<<axpytemp.y<<std::endl;
	
	hipDeviceSynchronize();

	if (linalgstat != HIPBLAS_STATUS_SUCCESS){
		std::cout<<"V1 = V1 - alpha*V0 failed! Error: ";
		std::cout<<linalgstat<<std::endl;
	}

	if (hipPeekAtLastError() != 0 ){
		std::cout<<"Getting V1  = V1 - a*V0 failed! Error: ";
		std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
	} 


	std::ofstream fout;
	fout.open("lanczos.log");
	//fout<<normtemp<<std::endl;
	fout<<std::endl;
	//int* h_H_vals = (int*)malloc((dim+1)*sizeof(int));
	hipMemcpy(host_v0, v1, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	for(int i = 0; i < dim ; i++){
		fout<<host_v0[i].x<<std::endl;
	}

	fout.close();

	
  
	
	hipEventRecord(start, 0);
	linalgstat = hipblasDznrm2(linalghandle, dim, v1, 1, &normtemp); //this is slow for some reason
  
	hipDeviceSynchronize();
	if (linalgstat != HIPBLAS_STATUS_SUCCESS){
		std::cout<<"Getting the norm of v1 failed! Error: ";
		std::cout<<linalgstat<<std::endl;
	}

	
	if (hipPeekAtLastError() != 0 ){
		std::cout<<"Getting nrm(V1) failed! Error: ";
		std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
	} 

 
	//d_b_ptr = thrust::raw_pointer_cast(&d_b[1]);

	h_b[1] = make_hipDoubleComplex(normtemp,0.);
	// this function (above) takes the norm
  	std::cout<<normtemp<<std::endl;
	normtemp = 1./normtemp;
	hipDoubleComplex gamma = make_hipDoubleComplex(1./hipCreal(h_b[1]),0.); //alpha = 1/beta in v1 = v1 - alpha*v0
  	//normalize<<<dim/512 + 1, 512>>>(v0, dim, normtemp);
	linalgstat = hipblasZdscal(linalghandle, dim, &normtemp, v1, 1);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	if (linalgstat != HIPBLAS_STATUS_SUCCESS){
		std::cout<<"Normalizing v1 failed! Error: ";
		std::cout<<linalgstat<<std::endl;
	}

	
	if (hipPeekAtLastError() != 0 ){
		std::cout<<"Normalizing V1 failed! Error: ";
		std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
	} 


	//std::cout<<"Time to normalize v1: "<<time<<std::endl;

	
	//Now we're done the first round!
	//*********************************************************************************************************

	/*thrust::device_vector<double> d_ordered(num_Eig);
	thrust::fill(d_ordered.begin(), d_ordered.end(), 0);
	double* d_ordered_ptr = thrust::raw_pointer_cast(&d_ordered[0]); */
	hipEventRecord(start, 0);
	double gs_Energy = 1.; //the lowest energy

	int returned;

	int iter = 0;

  // In the original code, we started diagonalizing from iter = 5 and above. I start from iter = 1 to minimize issues of control flow
  /*thrust::device_vector<double> d_diag(max_Iter);
  double* diag_ptr;
  thrust::device_vector<double> d_offdia(max_Iter);
  double* offdia_ptr;*/
  
	thrust::host_vector<double> h_diag(max_Iter);
	double* h_diag_ptr = raw_pointer_cast(&h_diag[0]);
	thrust::host_vector<double> h_offdia(max_Iter);
	double* h_offdia_ptr = raw_pointer_cast(&h_offdia[0]);
  
  thrust::device_vector<hipDoubleComplex> temp(dim);
  hipDoubleComplex* temp_ptr = thrust::raw_pointer_cast(&temp[0]);

  double eigtemp = 0.;

  thrust::host_vector<double> h_ordered(num_Eig, 0.);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//std::cout<<"Time to set up the arrays for iteration: "<<time<<std::endl;

  while( fabs(gs_Energy - eigtemp) > conv_req || iter < 10){ //this is a cleaner version than what was in the original - way fewer if statements

    iter++;

	eigtemp = h_ordered[num_Eig - 1];
    /*status1 = hipMemcpy(&eigtemp, d_ordered_ptr, sizeof(double), hipMemcpyDeviceToHost);

    if (status1 != hipSuccess){
      printf("Copying last eigenvalue failed \n");
    }*/
    //std::cout<<"Getting V2 = H*V1 for the "<<iter + 1<<"th time"<<std::endl;
		hipEventRecord(start, 0);
    sparsestatus = hipsparseZhybmv(sparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, H_descr, hyb_Ham, v1, &beta, v2); // the Hamiltonian is applied here, in this gross expression
    hipDeviceSynchronize();

		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to do csrmv: "<<time<<std::endl;

    if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
      std::cout<<"Error applying the Hamiltonian in "<<iter<<"th iteration!";
      std::cout<<"Error: "<<sparsestatus<<std::endl;
    } 

    //d_a_ptr = thrust::raw_pointer_cast(&d_a[iter]);
    //std::cout<<"Getting V1*V2 for the "<<iter + 1<<"th time"<<std::endl;
    
		hipEventRecord(start, 0);
		linalgstat = hipblasZdotc(linalghandle, dim, v1, 1, v2, 1, &dottemp);
    hipDeviceSynchronize();

		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to get v1*v2: "<<time<<std::endl;

    h_a[iter] = dottemp;

    if (linalgstat != HIPBLAS_STATUS_SUCCESS){
      std::cout<<"Error getting v1 * v2 in "<<iter<<"th iteration! Error: ";
      std::cout<<linalgstat<<std::endl;
    }

    //hipMemcpy(temp_ptr, v1, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    //temp = v1;

    axpytemp = hipCmul(make_hipDoubleComplex(-1., 0.), h_b[iter]);
    hipEventRecord(start, 0);
    linalgstat = hipblasZaxpy( linalghandle, dim, &axpytemp, v0, 1, v2, 1);
    if (linalgstat != HIPBLAS_STATUS_SUCCESS){
      std::cout<<"Error getting (d_b/d_a)*v0 + v1 in "<<iter<<"th iteration!";
      std::cout<<"Error: "<<linalgstat<<std::endl;
    }
    hipDeviceSynchronize();
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to get v2 - b[i]*v0: "<<time<<std::endl;

    axpytemp = hipCmul(make_hipDoubleComplex(-1., 0.), h_a[iter]);
    linalgstat = hipblasZaxpy( linalghandle, dim, &axpytemp, v1, 1, v2, 1);
    if (linalgstat != HIPBLAS_STATUS_SUCCESS){
      std::cout<<"Error getting v2 + d_a*v1 in "<<iter<<"th iteration! Error: ";
      std::cout<<linalgstat<<std::endl;
    }

    //std::cout<<"Getting norm of V2 for the "<<iter + 1<<"th time"<<std::endl;
		hipEventRecord(start, 0);
    linalgstat = hipblasDznrm2( linalghandle, dim, v2, 1, &normtemp);
    if (linalgstat != HIPBLAS_STATUS_SUCCESS){
      std::cout<<"Error getting norm of v2 in "<<iter<<"th iteration! Error: ";
      std::cout<<linalgstat<<std::endl;
    }

		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to get norm of v2: "<<time<<std::endl;

    h_b[iter + 1] = make_hipDoubleComplex(normtemp, 0.);
    gamma = make_hipDoubleComplex(1./normtemp,0.);

    linalgstat = hipblasZscal(linalghandle, dim, &gamma, v2, 1);
    if (linalgstat != HIPBLAS_STATUS_SUCCESS){ 
      std::cout<<"Error getting 1/d_b * v2 in "<<iter<<"th iteration! Error: ";
      std::cout<<linalgstat<<std::endl;
    }

		hipEventRecord(start, 0);
	//lancz_ptr = raw_pointer_cast(&d_lanczvec[dim*(iter - 1)]);
	//hipMemcpy(lancz_ptr, v0, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	hipMemcpy(v0, v1, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	hipMemcpy(v1, v2, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to copy around the Lanczos vectors: "<<time<<std::endl;
    
    for (int i = 0; i <= iter; i++){
			h_diag[i] = hipCreal(h_a[i]); //adding another spot in the tridiagonal matrix representation
    	h_offdia[i] = hipCreal(h_b[i]);
		}
		
  //this tqli stuff is a bunch of crap and needs to be fixed  
    //double* d_H_eigen;
    //size_t d_eig_pitch;

    /*status1 = hipMalloc(&d_H_eigen, max_Iter*max_Iter*sizeof(double));
    if (status1 != hipSuccess){
      printf("tqli eigenvectors matrix memory allocation failed! \n");
    }
    
    zero<<<(iter*iter)/512 + 1, 512>>>(d_H_eigen, iter*iter);
    unitdiag<<<iter/512 + 1, 512>>>(d_H_eigen, iter); //set this matrix to the identity */
    //h_diag = d_diag;
    //h_offdia = d_offdia;

    double* h_H_eigen = (double*)malloc(max_Iter*max_Iter*sizeof(double));
    //hipMemcpy(h_H_eigen, d_H_eigen, max_Iter*max_Iter*sizeof(double), hipMemcpyDeviceToHost);
    for (int ii=1;ii<=iter;ii++){
        h_offdia[ii-1] = h_offdia[ii];
    }
    h_offdia[iter] = 0;
		hipEventRecord(start, 0);
    		returned = tqli(h_diag_ptr, h_offdia_ptr, iter + 1, max_Iter, h_H_eigen); //tqli is in a separate file   
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Time to run TQLI: "<<time<<std::endl;
//
    //d_diag = h_diag;
    
		hipEventRecord(start, 0);
		thrust::sort(h_diag.begin(), h_diag.end());
    thrust::copy(h_diag.begin(), h_diag.begin() + num_Eig, h_ordered.begin());
		hipEventRecord(stop, 0);		
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		//std::cout<<"Runtime for sort and copy: "<<time<<std::endl;

		//std::sort(h_diag.begin(), h_diag.end());
   
	gs_Energy = h_ordered[num_Eig - 1];
    //d_ordered_ptr = thrust::raw_pointer_cast(&d_ordered[num_Eig - 1]);
    //status2 = hipMemcpy(&gs_Energy, d_ordered_ptr, sizeof(double), hipMemcpyDeviceToHost);
    
    /*h_ordered = d_ordered;

		if (status2 != hipSuccess){
      printf("Copying the eigenvalue failed! \n");
    }*/

	for(int i = 0; i < num_Eig; i++){
   		std::cout<<std::setprecision(12)<<h_ordered[i]<<" ";
  	} 
	std::cout<<std::endl;


    if (iter == max_Iter - 2){// have to use this or d_b will overflow
      //this stuff here is used to resize the main arrays in the case that we aren't converging quickly enough
      h_a.resize(2*max_Iter);
      h_b.resize(2*max_Iter);
      //d_diag.resize(2*max_Iter);
      //d_offdia.resize(2*max_Iter);
      h_diag.resize(2*max_Iter);
      h_offdia.resize(2*max_Iter);
      d_lanczvec.resize(2*max_Iter*dim);
      max_Iter *= 2;
    }
    //hipFree(d_H_eigen);
       
  } 

  
  for(int i = 0; i < num_Eig; i++){
    std::cout<<h_ordered[i]<<" ";
  } //write out the eigenenergies
  std::cout<<std::endl;
  // call the expectation values function
  
  // time to copy back all the eigenvectors
  //thrust::host_vector<hipDoubleComplex> h_lanczvec(max_Iter*dim);
  //h_lanczvec = d_lanczvec;
  
  // now the eigenvectors are available on the host CPU

  linalgstat = hipblasDestroy(linalghandle);
	
  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    printf("CUBLAS failed to shut down properly! \n");
  }

  sparsestatus = hipsparseDestroy(sparsehandle);

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    printf("CUSPARSE failed to release handle! \n");
  }
  hipFree(v0);
  hipFree(v1);
  hipFree(v2);

	hipEventDestroy(start);
	hipEventDestroy(stop);

}
// things left to do:
// write a thing (separate file) to call routines to find expectation values, should be faster on GPU 
// make the tqli thing better!

__global__ void normalize(hipDoubleComplex* v, const int size, double norm){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < size){
    v[i] = hipCdiv(v[i], make_hipDoubleComplex(norm, 0. ));
  }
}

int tqli(double* d, double* e, int n, int max_Iter, double *z)

{
 
  int m,l,iter,i,k;
  double s,r,p,g,f,dd,c,b;

  for (l=0;l<n;l++) {
    iter=0;
    do { 
      for (m=l;m<n-1;m++) { 
	dd=fabs(d[m])+fabs(d[m+1]);
	if (fabs(e[m])+dd == dd) break;
      }
      if (m!=l) { 
	if (iter++ == 30) { 
	  std::cout <<"Too many iterations in tqli() \n";
	  return 0;
	}
	g=(d[l+1]-d[l])/(2.0*e[l]);
	r=sqrt((g*g)+1.0);
	g=d[m]-d[l]+e[l]/(g+SIGN(r,g));
	s=c=1.0;
	p=0.0;
	for (i=m-1;i>=l;i--) { 
	  f=s*e[i];
	  b=c*e[i];
	  if (fabs(f) >= fabs(g)) { 
	    c=g/f;r=sqrt((c*c)+1.0);
	    e[i + 1]=f*r;
	    c *= (s=1.0/r);
	  }
	  else { 
	    s=f/g;r=sqrt((s*s)+1.0);
	    e[i+1]=g*r;
	    s *= (c=1.0/r);
	  }
	  g=d[i+1]-p;
	  r=(d[i]-g)*s+2.0*c*b;
	  p=s*r;
	  d[i+1]=g+p;
	  g=c*r-b;
	  /*EVECTS*/
	  /*
	    for (k=0;k<n;k++) { 
	      f=z(k,i+1);
	      z(k,i+1)=s*z(k,i)+c*f;
	      z(k,i)=c*z(k,i)-s*f;
	    }
	  */
	}
	d[l]=d[l]-p;
	e[l]=g;
	e[m]=0.0;
      }
    } while (m!=l);
  }
  return 1;
}


double pythag(double a, double b){
  double absa, absb;
  absa=fabs(a);
  absb=fabs(b);
  if (absa > absb) return absa*sqrt(1.0+(absb/absa)*(absb/absa));
  else return (absb == 0.0 ? 0.0 : absb*sqrt(1.0+(absa/absb)*(absa/absb)));
}
