#include "hip/hip_runtime.h"
// Katharine Hyatt
// A set of functions to implement the Lanczos method for a generic Hamiltonian
// Based on the codes Lanczos_07.cpp and Lanczos07.h by Roger Melko
//-------------------------------------------------------------------------------

#include"lanczos.h"

// h_ means this variable is going to be on the host (CPU)
// d_ means this variable is going to be on the device (GPU)
// s_ means this variable is shared between threads on the GPU
// The notation <<x,y>> before a function defined as global tells the GPU how many threads per block to use
// and how many blocks per grid to use
// blah.x means the real part of blah, if blah is a data type from hip/hip_complex.h
// blah.y means the imaginary party of blah, if blah is a data type from hip/hip_complex.h
// threadIdx.x (or block) means the "x"th thread from the left in the block (or grid)
// threadIdx.y (or block) means the "y"th thread from the top in the block (or grid)

//Function vecdiff: calculates the difference between some vectors, two of which is multiplied by a scalar
//Implements w = x - a*y - b*z 
//-------------------------------------------------------------------------------------------------------
//Input: w, a "dummy pointer" to the vector that is changed
//       x, the vector that a*y and b*z are subtracted from 
//       alpha, the scalar the first subtracted vector is multiplied by
//       y, the first subtracted vector
//       beta, the scalar the second subtraced vector is multiplied by
//       z, the second subtracted vector
//       n, the number of elements in all the vectors
//       Note: no control is put in here to make sure all vectors are the same size. 
//------------------------------------------------------------------------------------------------------
//Output: w, the result of the subtractions
//        All other quantities remain unchanged
//------------------------------------------------------------------------------------------------------
__global__ void vecdiff(hipDoubleComplex* w, hipDoubleComplex* x, hipDoubleComplex alpha, hipDoubleComplex* y, hipDoubleComplex beta, hipDoubleComplex* z, int n){

  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n) {
    w[i] = hipCsub(x[i],hipCsub(hipCmul(alpha,y[i]),hipCmul(beta,z[i]))); //this is the dirtiest thing ever
  }
  __syncthreads();
}

//Function assignr: assigns the real parts of an array of double complex numbers the value of some double
//-------------------------------------------------------------------------------------------------------
//Input: a, a vector of double precision complex numbers whose real parts we would like to change
//       b, the real number that will become the real part of the complex numbers in a
//       n, the number of elements in a
//-------------------------------------------------------------------------------------------------------
//Output: a, the vector of complex numbers whose real parts have been changed
//        All other quantities are unchanged
//------------------------------------------------------------------------------------------------------- 
__global__ void assignr(hipDoubleComplex* a, double b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n){
    a[i] = make_hipDoubleComplex(b,0.);
  }
}

__global__ void assign(double* a, double b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if (i < n){
    a[i] = b;
  }
}


//Function complextodoubler: assigns the real parts of complex numbers in an array to doubles in another array
//------------------------------------------------------------------------------------------------------------
//Input: a, the vector of complex numbers whose real parts we are extracting
//       b, the vector of doubles that will hold the real parts
//       n, the number of elements in the vectors
//------------------------------------------------------------------------------------------------------------
//Output: b, the vector of doubles now holding the real parts
//        All other quantities are unchanged
//------------------------------------------------------------------------------------------------------------
__global__ void complextodoubler(hipDoubleComplex* a, double* b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if(i <= n){
    b[i] = a[i].x; 
  }
}

//Same as above, but in this case the parts are shifted by one space in the vector
__global__ void complextodoubler2(hipDoubleComplex* a, double* b, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x + 1;

  if(i <= n){
    b[i-1] = a[i].x;
  }
  if(i == n+1){
    b[i-1] = 0.;
  }
} 

__global__ void zero(double** a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int j = blockDim.y*blockIdx.y + threadIdx.y;

  if ( i< m && j < m){
    a[i][j] = 0. ;
  }
}
// Note: to get the identity matrix, apply the fuction zero above first
__global__ void eye(double** a, int m){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (i < m ){

    a[i][i] = 1.;
  }
}


//Function copyHamiltonian: copies all the CSR data for the Hamiltonian to the device - made this to clean stuff up
__host__ void copyHamiltonian(const int h_num_nonzeroelem, const hipDoubleComplex* h_values, const int* h_rowstart, const int* h_colindex, const hipsparseMatDescr_t* h_descrH, int dim, int* d_num_nonzeroelem, hipDoubleComplex* d_values, int* d_rowstart, int* d_colindex, hipsparseMatDescr_t* d_descrH, int* d_dim){
  

  hipError_t error;

  error = hipMalloc(&d_num_nonzeroelem, sizeof(int));

  if (error != hipSuccess){
    printf("Num of elements allocation on device failed! \n");
  }

  error = hipMemcpy(d_num_nonzeroelem, &h_num_nonzeroelem, sizeof(int), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Num of elements copy from host to device failed! \n");
  }

  error = hipMalloc(&d_values, h_num_nonzeroelem*sizeof(hipDoubleComplex));

  if (error != hipSuccess){
    printf("Values allocation on device failed! \n");
  }

  error = hipMemcpy(d_values, h_values, h_num_nonzeroelem*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Values copy from host to device failed! \n");
  }

  error = hipMalloc(&d_rowstart, sizeof(h_rowstart)*sizeof(int));
  
  if (error != hipSuccess){
    printf("Row start allocation on device failed! \n");
  }

  error = hipMemcpy(d_rowstart, h_rowstart, sizeof(h_rowstart)*sizeof(int), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Row start copy from host to device failed! \n");
  }

  error = hipMalloc(&d_colindex, h_num_nonzeroelem*sizeof(int));

  if (error != hipSuccess){
    printf("Column index allocation on device failed! \n");
  }

  error = hipMemcpy(d_colindex, h_colindex, h_num_nonzeroelem*sizeof(int), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Column index copy from host to device failed! \n");
  }

  error = hipMalloc(&d_descrH, sizeof(hipsparseMatDescr_t));
  
  if (error != hipSuccess){
    printf("Matrix description allocation on device failed! \n");
  }

  error = hipMemcpy(d_descrH, h_descrH, sizeof(hipsparseMatDescr_t), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Matrix description copy from host to device failed! \n");
  }

  error = hipMalloc(&d_dim, sizeof(int));
  if (error != hipSuccess){
    printf("Matrix dimension allocation on device failed! \n");
  }

  error = hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);

  if (error != hipSuccess){
    printf("Matrix dimension copy from host to device failed! \n");
  }

}
//Function lanczos: takes a hermitian matrix H, tridiagonalizes it, and finds the n smallest eigenvalues.
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Input: num_nonzeroelem, the number of nonzero elements in the Hamiltonian
//        values, an array of the nonzero values
//        rowstart, an array of the index of the values that lead a row (see CSR matrix representation documentation for more on this)
//        colindex, and array of the column in which each element in values resides
//        descrH, the description of H that cusparse functions need
//        dim, the dimension of the full mxm matrix
//        max_Iter, the starting number of iterations we'll try
//        num_Eig, the number of eigenvalues we're interested in seeing
//        conv_req, the convergence we'd like to see
//---------------------------------------------------------------------------------------------------------------------------------------------------
// Output: h_ordered, the array of the num_Eig smallest eigenvalues, ordered from smallest to largest
//---------------------------------------------------------------------------------------------------------------------------------------------------        



int main(){}

void lanczos(const int h_num_nonzeroelem, const hipDoubleComplex* h_values, const int* h_rowstart, const int* h_colindex, const hipsparseMatDescr_t *h_descrH, const int dim, int max_Iter, const int num_Eig, const double conv_req){

  hipblasStatus_t linalgstat;
  linalgstat = cublasInit(); //have to initialize the cuBLAS environment, or my program won't work! I could use this later to check for errors as well

  hipsparseHandle_t sparsehandle;
  hipsparseStatus_t sparsestatus = hipsparseCreate(&sparsehandle); //have to initialize the cusparse environment too! This variable gets passed to all my cusparse functions

  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    printf("Failed to initialize CUBLAS! \n");
  }

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    printf("Failed to initialize CUSPARSE! \n");
  }

  hipError_t status1, status2, status3, status4; //this is to throw errors in case things (mostly memory) in the code fail!  

  int* d_num_nonzeroelem;
  hipDoubleComplex* d_values;
  int* d_rowstart;
  int* d_colindex; 
  hipsparseMatDescr_t* d_descrH;
  int* d_dim;

  copyHamiltonian(h_num_nonzeroelem, h_values, h_rowstart, h_colindex, h_descrH, dim, d_num_nonzeroelem, d_values, d_rowstart, d_colindex, d_descrH, d_dim);

  hipDoubleComplex* d_a; //these are going to store the elements of the tridiagonal matrix
  hipDoubleComplex* d_b; //they have to be hipDoubleComplex because that's the only input type the cublas functions I need will take

  status3 = hipMalloc(&d_a, max_Iter*sizeof(hipDoubleComplex));
  status4 = hipMalloc(&d_b, max_Iter*sizeof(hipDoubleComplex));

  if ((status3 != hipSuccess) || (status4 != hipSuccess)){
    printf("Matrix elements memory allocation failed! \n");
  }


  int tpb = 256; //threads per block - a conventional number
  int bpg = (dim + tpb - 1)/tpb; //blocks per grid

  //Making the "random" starting vector

  hipDoubleComplex** d_eigen_Array; //this thing is an array of pointers to the eigenvectors 
  status1 = hipMalloc(&d_eigen_Array, max_Iter*sizeof(hipDoubleComplex*)); // making the pointer array

  if (status1 != hipSuccess){
    printf("Eigenvector array allocation failed! \n");
  }
	//need to fix the below too
  CDCarraysalloc<<<1, max_Iter>>>(d_eigen_Array, dim, 0); //time to make the actual arrays of the eigenvectors

  assignr<<<bpg,tpb>>>(d_eigen_Array[0], 1., dim); //assigning the values of the "random" starting vector
  
  hipDoubleComplex* alpha; 
  status3 = hipMalloc(&alpha, sizeof(hipDoubleComplex));
  *alpha = make_hipDoubleComplex(1.,0.);
  hipDoubleComplex* beta;
  status4 = hipMalloc(&beta, sizeof(hipDoubleComplex));
  *beta = make_hipDoubleComplex(0.,0.);

  if ((status3 != hipSuccess) || (status4 != hipSuccess)){
    printf("Dummy constants alpha and beta initialization failed! \n");
  }

  hipsparseOperation_t* A;
  hipMalloc(&A, sizeof(hipsparseOperation_t));

  *A = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  hipsparseZcsrmv(sparsehandle, *A, *d_dim, *d_dim, *alpha, *d_descrH, d_values, d_rowstart, d_colindex, d_eigen_Array[0], *beta, d_eigen_Array[1]); // the Hamiltonian is applied here

  //*********************************************************************************************************
  // This is just the first steps so I can do the rest  
  d_a[0] = hipblasZdotc(*d_dim, d_eigen_Array[0], sizeof(hipDoubleComplex), d_eigen_Array[1], sizeof(hipDoubleComplex));
  d_b[0] = make_hipDoubleComplex(0.,0.);

  hipDoubleComplex* y;
  status2 = hipMalloc(&y, dim*sizeof(hipDoubleComplex));

  if (status2 != hipSuccess){
    printf("Memory allocation of y dummy vector failed! \n");
  }
  
  assignr<<<bpg,tpb>>>(y, 0., *d_dim); //a dummy vector of 0s that i can stick in my functions

  vecdiff<<<bpg,tpb>>>(d_eigen_Array[1], d_eigen_Array[1], d_a[0], d_eigen_Array[0], y[0], y, *d_dim);
  d_b[1] = make_hipDoubleComplex(sqrt(hipblasDznrm2(dim, d_eigen_Array[1], sizeof(hipDoubleComplex))),0.);
  // this function (above) takes the norm
  
  hipDoubleComplex gamma = make_hipDoubleComplex(1./d_b[1].x,0.); //alpha = 1/beta in v1 = v1 - alpha*v0

  hipblasZaxpy(dim, gamma, d_eigen_Array[1], sizeof(hipDoubleComplex), y, sizeof(hipDoubleComplex)); // function performs a*x + y

  //Now we're done the first round!
  //*********************************************************************************************************

  double* d_ordered;
  status1 = hipMalloc(&d_ordered, num_Eig*sizeof(double));

  if (status1 != hipSuccess){
    printf("Eigenvalue array memory allocation failed! \n");
  }

  assign<<<bpg,tpb>>>(d_ordered, 0., num_Eig);

  double* gs_Energy;
  *gs_Energy = 1.; //the lowest energy

  int returned;

  int iter = 0;

  // In the original code, we started diagonalizing from iter = 5 and above. I start from iter = 1 to minimize issues of control flow
  double* d_diag;
  double* d_offdia;

  status3 = hipMalloc(&d_diag, dim*sizeof(double));
  status4 = hipMalloc(&d_offdia, dim*sizeof(double));

  thrust::device_ptr<double> dev_ptr(d_diag);

  if ((status3 != hipSuccess) || (status4 != hipSuccess)){
    printf("Second matrix elements array memory allocation failed! \n");
  }

  double* eigtemp;
  *eigtemp = 0.;

  while( abs(*gs_Energy - *eigtemp)> conv_req){ //this is a cleaner version than what was in the original - way fewer if statements

    iter++;

    status1 = hipMemcpy(eigtemp, &d_ordered[num_Eig - 1], sizeof(double), hipMemcpyDeviceToHost);

    if (status1 != hipSuccess){
      printf("Copying last eigenvalue failed \n");
    }

    hipsparseZcsrmv(sparsehandle, *A, *d_dim, *d_dim, *alpha, *d_descrH, d_values, d_rowstart, d_colindex, d_eigen_Array[iter], *beta, d_eigen_Array[iter+1]); // the Hamiltonian is applied here, in this gross expression

    d_a[iter] = hipblasZdotc(*d_dim, d_eigen_Array[iter], sizeof(hipDoubleComplex), d_eigen_Array[iter + 1], sizeof(hipDoubleComplex));

    vecdiff<<<bpg,tpb>>>(d_eigen_Array[iter+1], d_eigen_Array[iter+1], d_a[iter], d_eigen_Array[iter], d_b[iter], d_eigen_Array[iter - 1], dim);

    d_b[iter+1] = make_hipDoubleComplex(sqrt(hipblasDznrm2(dim, d_eigen_Array[iter+1], sizeof(hipDoubleComplex))),0.);
    
    hipDoubleComplex* gamma;
    hipMalloc(&gamma, sizeof(hipDoubleComplex));
    *gamma = make_hipDoubleComplex(1./d_b[iter+1].x,0.);
    hipblasZaxpy(d_dim, gamma, d_eigen_Array[iter+1], sizeof(hipDoubleComplex), y, sizeof(hipDoubleComplex));
    
    //hipblasZcopy(dim, d_v_Mid, sizeof(hipDoubleComplex), d_v_Start, sizeof(hipDoubleComplex)); //switching my vectors around for the next iteration
    //hipblasZcopy(dim, d_v_End, sizeof(hipDoubleComplex), d_v_Mid, sizeof(hipDoubleComplex)); unnecesarry now that i'm using the array of pointers

    d_diag[iter] = 0.; //adding another spot in the tridiagonal matrix representation
    d_offdia[iter] = 0.;

    complextodoubler<<<bpg,tpb>>>(d_a, d_diag, iter);
    complextodoubler2<<<bpg,tpb>>>(d_b, d_offdia, iter);

    double** d_H_eigen;
    size_t d_eig_pitch;

    status1 = hipMallocPitch(&d_H_eigen, &d_eig_pitch, iter*sizeof(double), iter);
    if (status1 != hipSuccess){
      printf("tqli eigenvectors matrix memory allocation failed! \n");
    }
    
    zero<<<bpg,tpb>>>(d_H_eigen, iter);
    eye<<<bpg,tpb>>>(d_H_eigen, iter); //set this matrix to the identity

    returned = tqli(d_diag, d_offdia, iter + 1, d_H_eigen); //tqli is in a separate file   

    //assign<<<tpb,bpg>>>(d_ordered, d_diag[0], num_Eig);
    
    
    thrust::sort(dev_ptr, dev_ptr + *d_dim); //sorts the array of eigenvalues    

    hipMemcpy(d_ordered, d_diag, num_Eig*sizeof(double), hipMemcpyDeviceToDevice);


    status2 = hipMemcpy(&gs_Energy, &(d_ordered[num_Eig - 1]), sizeof(double), hipMemcpyDeviceToHost);

    if (status2 != hipSuccess){
      printf("Copying the eigenvalue failed! \n");
    }

    if (iter == sizeof(d_eigen_Array) - 2){// have to use this or d_b will overflow
      //this stuff here is used to resize the main arrays in the case that we aren't converging quickly enough
      //------------------------------------------------------------------------
	hipDoubleComplex* temp;
        status1 = hipMalloc(&temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        
        status2 = hipMemcpy(temp, d_a, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
        
        hipFree(d_a);
        status3 = hipMalloc(&d_a, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        status4 = hipMemcpy(d_a, temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
        
        if( (status1 != hipSuccess) ||
            (status2 != hipSuccess) ||
            (status3 != hipSuccess) ||
            (status4 != hipSuccess) ){
          printf("Resizing d_a failed! \n");
        } 

        status1 = hipMemcpy(temp, d_b, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

        hipFree(d_b);
        status2 = hipMalloc(&d_b, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex));
        status3 = hipMemcpy(d_b, temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

        if ( (status1 != hipSuccess) ||
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ){
          printf("Resizing d_b failed! \n");
        }
          
        hipFree(temp);
        hipMalloc(&temp, (2*sizeof(d_eigen_Array) + 1)*sizeof(hipDoubleComplex*));
        
        status1 = hipMemcpy(temp, d_eigen_Array, sizeof(d_eigen_Array)*sizeof(hipDoubleComplex*), hipMemcpyDeviceToDevice);

        int temp2 = sizeof(d_eigen_Array);

        status2 = hipFree(d_eigen_Array);
        status3 = hipMalloc(&d_eigen_Array, (2*temp2 + 1)*sizeof(hipDoubleComplex*));
        status4 = hipMemcpy(d_eigen_Array, temp, (2*temp2 + 1)*sizeof(hipDoubleComplex*), hipMemcpyDeviceToDevice);

        if ( (status1 != hipSuccess) ||
             (status2 != hipSuccess) ||
             (status3 != hipSuccess) ||
             (status4 != hipSuccess) ){
          printf("Resizing d_eigen_Array failed! \n");
        }

        CDCarraysalloc<<<1, temp2 + 1>>>(d_eigen_Array, dim, temp2);//need to change this
                
        hipFree(temp); 
        //resizing

    }   
  } 

  double* h_ordered;

  status1 = hipHostMalloc(&h_ordered, num_Eig*sizeof(double)); //a place to put the eigenvalues on the CPU

  if (status1 != hipSuccess){
    printf("Memory allocation for host eigenvector array failed! \n");
  }

  status2 = hipMemcpy(h_ordered, d_ordered, num_Eig*sizeof(double), hipMemcpyDeviceToHost); // moving the eigenvalues over

  if (status2 != hipSuccess){
    printf("Copying eigenvalues from GPU to CPU failed! \n");
  }

  for(int i = 0; i < num_Eig; i++){
    printf("%lf \n", h_ordered[i]);
  } //write out the eigenenergies

  hipFree(&alpha);
  hipFree(&beta);
  hipFree(d_a);
  hipFree(d_b); //dropping stuff off
  // call the expectation values function
  
  // time to copy back all the eigenvectors
  //int* sizeptr;
  //hipMemcpy(sizeptr, &sizeof(d_eigen_Array), sizeof(int), hipMemcpyDeviceToHost);
  
  max_Iter = sizeof(d_eigen_Array);

  hipDoubleComplex** h_eigen_Array;
  status1=hipHostMalloc(&h_eigen_Array, max_Iter*sizeof(hipDoubleComplex*));

  if (status1 != hipSuccess){
    printf("CPU eigenvector array memory allocation failed! \n");
  }
  
  for(int i = 0; i < max_Iter; i++){
     status2 = hipHostMalloc(&h_eigen_Array[i], dim*sizeof(hipDoubleComplex*));
     status3 = hipMemcpy(h_eigen_Array[i], d_eigen_Array[i], dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
     if ( (status2 != hipSuccess) || (status3 != hipSuccess)){
       printf("Copying an eigenvector array failed! \n");
     }

  } // now the eigenvectors are available on the host CPU

  linalgstat = cublasShutdown();
	
  if (linalgstat != HIPBLAS_STATUS_SUCCESS){
    printf("CUBLAS failed to shut down properly! \n");
  }

  sparsestatus = hipsparseDestroy(sparsehandle);

  if (sparsestatus != HIPSPARSE_STATUS_SUCCESS){
    printf("CUSPARSE failed to release handle! \n");
  }
}
// things left to do:
// write a thing (separate file) to call routines to find expectation values, should be faster on GPU 
// make the tqli thing better!
// do the hamiltonian generating stuff
// change things in here to set device array values properly

