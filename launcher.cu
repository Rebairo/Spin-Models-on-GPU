#include"lattice.h"
#include<cstdlib>
#include"hip/hip_runtime.h"
#include<iostream>
#include"lanczos.h"
//#include"hamiltonian.h"
int main()
{
    for(int i = 0; i < 1; i++)
    {
        int** Bond;
        //cout<<i<<" "<<endl;
        int how_many = 1;
        /*if (i == 1)
        {
            how_many = 5;
        }*/

        Bond = (int**)malloc(how_many*sizeof(int*));

        d_hamiltonian* hamil_lancz = (d_hamiltonian*)malloc(how_many*sizeof(d_hamiltonian));

        parameters* data = (parameters*)malloc(how_many*sizeof(parameters));

        double** groundstates = (double**)malloc(how_many*sizeof(double*));

        if (data == NULL)
        {
            cerr<<"Malloc of parameter container failed!"<<endl;
            return 1;
        }

        int* num_Elem = (int*)malloc(how_many*sizeof(int));

        //hipSetDevice(1);
        int device = 0; //i%2;

        for(int i = 0; i < how_many; i++)
        {

            data[i].nsite = 16;
            Bond[i] = (int*)malloc(3*data[i].nsite*sizeof(int));
            //Fill_Bonds_16B(Bond[i]);
            for( int j = 0; j < data[i].nsite; j++ ){
              Bond[i][j] = j;
              Bond[i][j+ data[i].nsite] = (j+1)%data[i].nsite;
            }
            data[i].Sz = 0;
            data[i].J1 = 4.f;
            data[i].J2 = 0.f;
            data[i].model_type = 2;
        }


        /*hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);*/
        //float time;

        //hipEventRecord(start,0);
        ConstructSparseMatrix(how_many, Bond, hamil_lancz, data, num_Elem, device);
        /*hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cout<<"Time to construct Hamiltonians: "<<time<<endl;
        hipEventRecord(start,0);
        */
        lanczos(how_many, num_Elem, hamil_lancz, groundstates, 200, 3, 1e-12);
        /*
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cout<<"Time to perform Lanczos: "<<time<<endl;*/
        for(int j = 0; j<how_many; j++)
        {
            hipFree(hamil_lancz[j].rows);
            hipFree(hamil_lancz[j].cols);
            hipFree(hamil_lancz[j].vals);
            hipFree(groundstates[j]);
        }
        //hipEventDestroy(start);
        //hipEventDestroy(stop);
        free(data);
        free(Bond);
        free(hamil_lancz);
        free(num_Elem);
        free(groundstates);
    }
    return 0;
}
