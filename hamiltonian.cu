#include "hip/hip_runtime.h"
#include"hamiltonian.h"

/* NOTE: this function uses FORTRAN style matrices, where the values and positions are stored in a ONE dimensional array! Don't forget this! */


int main(){


int* Bond;
  Bond = (int*)malloc(16*3*sizeof(int));

  Bond[0] = 0; Bond[1] = 1; Bond[2] = 2; Bond[3] = 3; Bond[4] = 4;
  Bond[5] = 5; Bond[6] = 6; Bond[7] = 7; Bond[8] = 8; Bond[9] = 9;
  Bond[10] = 10; Bond[11] = 11; Bond[12] = 12; Bond[13] = 13; Bond[14] = 14;
  Bond[15] = 15; Bond[16] = 1; Bond[17] = 2; Bond[18] = 3; Bond[19] = 0;
  Bond[20] = 5; Bond[21] = 6; Bond[22] = 7; Bond[23] = 4; Bond[24] = 9;
  Bond[25] = 10; Bond[26] = 11; Bond[27] = 8; Bond[28] = 13; Bond[29] = 14;
  Bond[30] = 15; Bond[31] = 12; Bond[32] = 4; Bond[33] = 5; Bond[34] = 6;
  Bond[35] = 7; Bond[36] = 8; Bond[37] = 9; Bond[38] = 10; Bond[39] = 11;
  Bond[40] = 12; Bond[41] = 13; Bond[42] = 14; Bond[43] = 15; Bond[44] = 0;
  Bond[45] = 1; Bond[46] = 2; Bond[47] = 3;

  hipDoubleComplex* hamil_Values;

  int* hamil_PosRow;

  int* hamil_PosCol;

  int nsite = 16;

  int dim;

  int Sz = 0;
  float JJ = 1.f;



  int num_Elem = ConstructSparseMatrix( 0, nsite, Bond, hamil_Values, hamil_PosRow, hamil_PosCol, &dim, JJ, Sz );

  return 0;
}

__host__ __device__ int idx(int i, int j, int lda){
  
  return (j + (i*lda));
}


/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
lattice_Size - the number of sites
Sz - the value of the Sz operator
basis_Position[] - an empty array that records the positions of the basis
basis - an empty array that records the basis
Outputs: basis_Position - a full array now
basis[] - a full array now

*/
__host__ int GetBasis(int dim, int lattice_Size, int Sz, int basis_Position[], int basis[]){
unsigned int temp = 0;
	int realdim = 0;

	for (unsigned int i1=0; i1<dim; i1++){
		temp = 0;
		basis_Position[i1] = -1;
		for (int sp =0; sp<lattice_Size; sp++){
			temp += (i1>>sp)&1;
		} //unpack bra
		//if (temp==(lattice_Size/2+Sz) ){
			basis[realdim] = i1;
			basis_Position[i1] = realdim;
			realdim++;
			//cout<<basis[realdim]<<" "<<basis_Position[i1]<<endl;
		//}
}

return realdim;

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
bra - the state
JJ - the coupling constant
Outputs: valH - the value of the Hamiltonian

*/

__device__ float HOffBondX(const int si, const int bra, const float JJ){

	float valH;
	//int S0, S1;
	//int T0, T1;

	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

	return valH;

}

__device__ float HOffBondY(const int si, const int bra, const float JJ){

	float valH;
	//int S0, S1;
	//int T0, T1;

	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

	return valH;


}

__device__ float HDiagPart(const int bra, int lattice_Size, int3* d_Bond, const float JJ){

	int S0b,S1b ; //spins (bra
	int T0,T1; //site
	//int P0, P1, P2, P3; //sites for plaquette (Q)
	//int s0p, s1p, s2p, s3p;
	float valH = 0.f;

	for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

		T0 = (d_Bond[Ti]).x; //lower left spin
		S0b = (bra>>T0)&1;
		//if (T0 != Ti) cout<<"Square error 3\n";
		T1 = (d_Bond[Ti]).y; //first bond
		S1b = (bra>>T1)&1; //unpack bra
		valH += JJ*(S0b-0.5)*(S1b-0.5);
		T1 = (d_Bond[Ti]).z; //second bond
		S1b = (bra>>T1)&1; //unpack bra
		valH += JJ*(S0b-0.5)*(S1b-0.5);

	}//T0

	//cout<<bra<<" "<<valH<<endl;

	return valH;

}//HdiagPart

/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
lattice_Size - the number of lattice sites
Bond - the bond values ??
hamil_Values - an empty pointer for a device array containing the values
hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs: hamil_Values - a pointer to a device array containing the values
hamil_PosRow - a pointer to a device array containing the locations of each value in a row
hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


__host__ int ConstructSparseMatrix(int model_Type, int lattice_Size, int* Bond, hipDoubleComplex* hamil_Values, int* hamil_PosRow, int* hamil_PosCol, int* vdim, float JJ, int Sz){


	//hipSetDevice(1);

	int num_Elem = 0; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	//int dim = 65536;

	/*
	switch (model_Type){
	case 0:
		dim = 65536;
		break;
	case 1: dim = 10; //guesses
	}
	*/
	int dim = 2;

	for (int ch=1; ch<lattice_Size; ch++) dim *= 2;

	int stride = 4*lattice_Size + 1;

	int basis_Position[dim];
	int basis[dim];
	//----------------Construct basis and copy it to the GPU --------------------//

	*vdim = GetBasis(dim, lattice_Size, Sz, basis_Position, basis);

	int* d_basis_Position;
	int* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(int));
	status2 = hipMalloc(&d_basis, *vdim*sizeof(int));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory allocation for basis arrays failed! Error: ";
		std::cout<<hipPeekAtLastError()<<std::endl;
		return 1;
	}

	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(int), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, *vdim*sizeof(int), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory copy for basis arrays failed! Error: ";
		std::cout<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	int* d_Bond;
	status1 = hipMalloc(&d_Bond, 3*lattice_Size*sizeof(int));

	status2 = hipMemcpy(d_Bond, Bond, 3*lattice_Size*sizeof(int), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		std::cout<<"Memory allocation and copy for bond data failed! Error: ";
		std::cout<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	int padded_dim = (*vdim/1024 + 1)*1024;
	int raw_size = (padded_dim + 4*lattice_Size*(*vdim));

	dim3 bpg;

	bpg.x = (4*lattice_Size*(*vdim))/512 + 1;
        
	dim3 tpb;
	tpb.x = 1024;
	//these are going to need to depend on dim and Nsize
     
	int* d_H_rows;
	int* d_H_cols;
	float* d_H_vals;
	
	hipMalloc(&d_H_rows, raw_size*sizeof(int));
	hipMalloc(&d_H_cols, raw_size*sizeof(int));
	hipMalloc(&d_H_vals, raw_size*sizeof(float));

	/*hamstruct* d_H_sort;
	status2 = hipMalloc(&d_H_sort, *vdim*stride*sizeof(hamstruct));

	if (status2 != hipSuccess){
		std::cout<<"Allocating d_H_sort failed! Error: ";
		std::cout<<hipGetErrorString( status1 )<<std::endl;
		return 1;
	}*/
	
	FillDiagonals<<<*vdim/1024 + 1, tpb>>>(d_basis, *vdim, d_H_rows, d_H_cols, d_H_vals, d_Bond, lattice_Size, JJ);

	hipDeviceSynchronize();

	if( hipPeekAtLastError() != 0 ){
		std::cout<<"Error in FillDiagonals! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, *vdim, d_H_rows, d_H_cols, d_H_vals, d_Bond, lattice_Size, JJ);

	hipDeviceSynchronize();

	if( hipPeekAtLastError() != 0 ){
		std::cout<<"Error in FillSparse! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	int* num_ptr;
	hipGetSymbolAddress((void**)&num_ptr, (const char*)"d_num_Elem");

	hipMemcpy(&num_Elem, num_ptr, sizeof(int), hipMemcpyDeviceToHost);
	std::cout<<num_Elem<<std::endl;
	status1 = hipFree(d_basis);
	status2 = hipFree(d_basis_Position);
	status3 = hipFree(d_Bond); // we don't need these later on

	if ( (status1 != hipSuccess) ||
			 (status2 != hipSuccess) ||
			 (status3 != hipSuccess) ){
		std::cout<<"Freeing bond and basis information failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	//----------------Sorting Hamiltonian--------------------------//

	sortEngine_t engine;
	sortStatus_t sortstatus = sortCreateEngine("sort/sort/src/cubin64/", &engine);

	MgpuSortData sortdata;

	
	sortdata.AttachKey((uint*)d_H_rows);
	sortdata.AttachVal(0, (uint*)d_H_cols);
	sortdata.AttachVal(1, (uint*)d_H_vals);

	int sortnumber = ((raw_size/2048) + 1)*2048;

	sortdata.Alloc(engine, sortnumber, 2);

	sortdata.firstBit = 0;
	sortdata.endBit = 8*sizeof(dim);

	sortArray(engine, &sortdata);

	/*thrust::device_ptr<int> sort_key_ptr(d_H_rows);
	thrust::device_ptr<int> sort_val_ptr(d_H_cols);

	thrust::sort_by_key(sort_key_ptr, sort_key_ptr + *vdim*stride, sort_val_ptr);*/
        
	//--------------------------------------------------------------

	if (hipPeekAtLastError() != 0){
		std::cout<<"Error in sorting! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&hamil_PosRow, num_Elem*sizeof(int));
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(int));

	if ( (status1 != hipSuccess) ||
			 (status2 != hipSuccess) ||
			 (status3 != hipSuccess) ){
		std::cout<<"Memory allocation for COO representation failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<std::endl;
		return 1;
	}

	hipMemcpy(hamil_PosRow, (int*)sortdata.keys[0], num_Elem*sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(hamil_PosCol, (int*)sortdata.values1[0], num_Elem*sizeof(int), hipMemcpyDeviceToDevice);

	FullToCOO<<<num_Elem/1024 + 1, 1024>>>(num_Elem, (float*)sortdata.values2[0], hamil_Values, *vdim); // csr and description initializations happen somewhere else

	hipFree(d_H_rows);
	hipFree(d_H_cols);
	hipFree(d_H_vals);

	hipDoubleComplex* h_vals = (hipDoubleComplex*)malloc(num_Elem*sizeof(hipDoubleComplex));
	int* h_rows = (int*)malloc(num_Elem*sizeof(int));
	int* h_cols = (int*)malloc(num_Elem*sizeof(int));

	/*hipMemcpy(h_vals, hamil_Values, num_Elem*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(h_rows, hamil_PosRow, num_Elem*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_cols, hamil_PosCol, num_Elem*sizeof(int), hipMemcpyDeviceToHost);

	std::ofstream fout;
	fout.open("hamiltonian.log");
	for(int i = 0; i < num_Elem; i++){
		fout<<"("<<h_rows[i]<<","<<h_cols[i]<<")";
		fout<<" - "<<h_vals[i].x<<std::endl;
	}

	fout.close();*/

	sortReleaseEngine(engine);

	return num_Elem;
}

__global__ void FillDiagonals(int* d_basis, int dim, int* H_rows, int* H_cols, float* H_vals, int* d_Bond, int lattice_Size, float JJ){

	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int site = threadIdx.x%(lattice_Size);

	unsigned int tempi = d_basis[row];

	__shared__ int3 tempbond[16];

	if (row < dim){
		(tempbond[site]).x = d_Bond[site];
		(tempbond[site]).y = d_Bond[lattice_Size + site];
		(tempbond[site]).z = d_Bond[2*lattice_Size + site];

		H_vals[row] = HDiagPart(tempi, lattice_Size, tempbond, JJ);
		H_rows[row] = row;
		H_cols[row] = row;

	}

	else {
		H_rows[row] = dim;
	}

}

/* Function FillSparse: this function takes the empty Hamiltonian arrays and fills them up. Each thread in x handles one ket |i>, and each thread in y handles one site T0
Inputs: d_basis_Position - position information about the basis
d_basis - other basis infos
d_dim - the number of kets
H_sort - an array that will store the Hamiltonian
d_Bond - the bond information
d_lattice_Size - the number of lattice sites
JJ - the coupling parameter

*/

__global__ void FillSparse(int* d_basis_Position, int* d_basis, int dim, int* H_rows, int* H_cols, float* H_vals, int* d_Bond, const int lattice_Size, const float JJ){

	int ii = (blockDim.x/(2*lattice_Size))*blockIdx.x + threadIdx.x/(2*lattice_Size);
	int T0 = threadIdx.x%(2*lattice_Size);

	#if __CUDA_ARCH__ < 200
		const int array_size = 512;
	#elif __CUDA_ARCH__ >= 200
		const int array_size = 1024;
	#else
       		#error your mom
	#endif

	__shared__ int3 tempbond[16];
	int count;
	__shared__ int temppos[array_size];
	__shared__ float tempval[array_size];
	__shared__ uint tempi[array_size];
	__shared__ uint tempod[array_size];

	int stride = 4*lattice_Size;
	int tempcount;
	int site = T0%(lattice_Size);
	count = 0;
	int rowtemp;

	int start = (dim/array_size + 1)*array_size;

	int si, sj;//sk,sl; //spin operators
	//unsigned int tempi;// tempod; //tempj;
	//hipDoubleComplex tempD;

	tempi[threadIdx.x] = d_basis[ii];

	__syncthreads();

	bool compare;

	if( ii < dim ){
		if (T0 < 2*lattice_Size){
			//Putting bond info in shared memory
			(tempbond[site]).x = d_Bond[site];
			(tempbond[site]).y = d_Bond[lattice_Size + site];
			(tempbond[site]).z = d_Bond[2*lattice_Size + site];

			__syncthreads();
			//Diagonal Part

			/*temppos[threadIdx.x] = d_basis_Position[tempi[threadIdx.x]];
			tempval[threadIdx.x] = HDiagPart(tempi[threadIdx.x], lattice_Size, tempbond, JJ);

			H_sort[ idx(ii, 0, stride) ].value = tempval[threadIdx.x];
			H_sort[ idx(ii, 0, stride) ].colindex = temppos[threadIdx.x];
			H_sort[ idx(ii, 0, stride) ].rowindex = ii;
			H_sort[ idx(ii, 0, stride) ].dim = dim;*/
                
			//-------------------------------
			//Horizontal bond ---------------
			si = (tempbond[site]).x;
			tempod[threadIdx.x] = tempi[threadIdx.x];
			sj = (tempbond[site]).y;

			tempod[threadIdx.x] ^= (1<<si); //toggle bit
			tempod[threadIdx.x] ^= (1<<sj); //toggle bit

			compare = (d_basis_Position[tempod[threadIdx.x]] > ii);
			temppos[threadIdx.x] = (compare) ? d_basis_Position[tempod[threadIdx.x]] : dim;
			tempval[threadIdx.x] = HOffBondX(site, tempi[threadIdx.x], JJ);

			count += (int)compare;
			tempcount = (T0/lattice_Size);
			rowtemp = (T0/lattice_Size) ? ii : temppos[threadIdx.x];			
			rowtemp = (compare) ? rowtemp : dim;

			H_vals[ idx(ii, 4*site + tempcount + start, stride) ] = tempval[threadIdx.x]; //(T0/lattice_Size) ? tempval[threadIdx.x] : hipConj(tempval[threadIdx.x]);
			H_cols[ idx(ii, 4*site + tempcount + start, stride) ] = (T0/lattice_Size) ? temppos[threadIdx.x] : ii;
			H_rows[ idx(ii, 4*site + tempcount + start, stride) ] = rowtemp;

//Vertical bond -----------------
			tempod[threadIdx.x] = tempi[threadIdx.x];
			sj = (tempbond[site]).z;

			tempod[threadIdx.x] ^= (1<<si); //toggle bit
			tempod[threadIdx.x] ^= (1<<sj); //toggle bit
                 
			compare = (d_basis_Position[tempod[threadIdx.x]] > ii);
			temppos[threadIdx.x] =  (compare) ? d_basis_Position[tempod[threadIdx.x]] : dim;
			tempval[threadIdx.x] = HOffBondY(site,tempi[threadIdx.x], JJ);

			count += (int)compare;
			tempcount = (T0/lattice_Size);
			rowtemp = (T0/lattice_Size) ? ii : temppos[threadIdx.x];			
			rowtemp = (compare) ? rowtemp : dim;

			H_vals[ idx(ii, 4*site + 2 + tempcount + start, stride) ] =  tempval[threadIdx.x]; // (T0/lattice_Size) ? tempval[threadIdx.x] : hipConj(tempval[threadIdx.x]);
			H_cols[ idx(ii, 4*site + 2 + tempcount + start, stride) ] = (T0/lattice_Size) ? temppos[threadIdx.x] : ii;
			H_rows[ idx(ii, 4*site + 2 + tempcount + start, stride) ] = rowtemp;
			
			__syncthreads();

			atomicAdd(&d_num_Elem, count);
}
}//end of ii
}//end of FillSparse

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
H_vals - the Hamiltonian values
H_pos - the Hamiltonian positions
hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(int num_Elem, float* H_vals, hipDoubleComplex* hamil_Values, int dim){

	int i = threadIdx.x + blockDim.x*blockIdx.x;

	if (i < num_Elem){

		hamil_Values[i].x = H_vals[i];
		

	}
}
;


