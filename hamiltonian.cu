#include "hip/hip_runtime.h"
#include"hamiltonian.h"


/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
	lattice_Size - the number of sites
	Sz - the value of the Sz operator
	basis_Position[] - an empty array that records the positions of the basis
	basis - an empty array that records the basis
Outputs:	basis_Position - a full array now
		basis[] - a full array now

*/
__host__ void GetBasis(int dim, int lattice_Size, int Sz, long basis_Position[], long basis[]){
	unsigned long temp = 0;

	for (unsigned long i1=0; i1<dim; i1++){
      		temp = 0;
		basis_Position[i1] = -1;
      		for (int sp =0; sp<lattice_Size; sp++){
          		temp += (i1>>sp)&1;
		}  //unpack bra
      		if (temp==(lattice_Size/2+Sz) ){ 
          		basis[i1] = i1;
          		basis_Position[i1] = i1 -1;
      		}
  	} 

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
        bra - the state
        JJ - the coupling constant
Outputs:  valH - the value of the Hamiltonian 

*/

__device__ hipDoubleComplex HOffBondX(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0.); //contribution from the J part of the Hamiltonian

  	return valH;


} 

__device__ hipDoubleComplex HOffBondY(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0. ); //contribution from the J part of the Hamiltonian

  	return valH;


}

__device__ hipDoubleComplex HDiagPart(const long bra, int lattice_Size, long* d_Bond, const double JJ){

  int S0b,S1b ;  //spins (bra 
  int T0,T1;  //site
  int P0, P1, P2, P3; //sites for plaquette (Q)
  int s0p, s1p, s2p, s3p;
  hipDoubleComplex valH = make_hipDoubleComplex( 0. , 0.);

  for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

    T0 = d_Bond[Ti]; //lower left spin
    S0b = (bra>>T0)&1;  
    //if (T0 != Ti) cout<<"Square error 3\n";
    T1 = d_Bond[Ti + lattice_Size]; //first bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);
    T1 = d_Bond[Ti + 2*lattice_Size]; //second bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);

  }//T0

  //cout<<bra<<" "<<valH<<endl;

  return valH;

}//HdiagPart 

/* Function: SetFirst - sets the first row elements of a matrix to some value

Inputs: H_pos - an array on the device whose first row elements will be changed
	dim - the number of rows in H_pos
	value - the value we want to set the first elements to
*/

__global__ void SetFirst(long** H_pos, long dim, long value){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < dim){
      
      H_pos[i][0] = value;
    }
}
/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
	lattice_Size - the number of lattice sites
	Bond - the bond values ??
	hamil_Values - an empty pointer for a device array containing the values 
	hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
	hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs:  hamil_Values - a pointer to a device array containing the values 
	hamil_PosRow - a pointer to a device array containing the locations of each value in a row
	hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


int ConstructSparseMatrix(int model_Type, int lattice_Size, long* Bond, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol){
	
	unsigned long num_Elem = 0; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	int dim;
	
	switch (model_Type){
		case 0: dim = 2;
		case 1: dim = 10; //guesses
	}

	
	long* d_Bond;
	status1 = hipMalloc(&d_Bond, 3*lattice_Size*sizeof(long));

	status2 = hipMemcpy(d_Bond, Bond, 3*lattice_Size*sizeof(long), hipMemcpyHostToDevice);

	status1 = hipPeekAtLastError();

	/*if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation and copy for bond data failed!";
		return 1;
	}*/

	if (status1 != hipSuccess){
		cout<<"Bond data allocation error: "<<hipGetErrorString( status1 )<<endl;
		return 1;
	}

	if (status2 != hipSuccess){
		cout<<"Bond data copy error: "<<hipGetErrorString( status2 )<<endl;
		return 1;
	}


	for (int ch=1; ch<lattice_Size; ch++) dim *=2;

        
	long basis_Position[dim];
	long basis[dim];

	int Sz = 0;

	//----------------Construct basis and copy it to the GPU --------------------//
	GetBasis(dim, lattice_Size, Sz, basis_Position, basis);

	long* d_basis_Position;
	long* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(long));
	status2 = hipMalloc(&d_basis, dim*sizeof(long));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation for basis arrays failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(long), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, dim*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory copy for basis arrays failed! Error: ";
                cout<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}

	dim3 bpg = (dim/256, 16, 1);
	dim3 tpb = (256, 1, 1); //these are going to need to depend on dim and Nsize

	//--------------Declare the Hamiltonian arrays on the device, and copy the pointers to them to the device -----------//

	long** h_H_pos;
	hipDoubleComplex** h_H_vals; 

	//status1 = hipMalloc(&H_pos, dim*sizeof(long*));
	//status2 = hipMalloc(&H_vals, dim*sizeof(hipDoubleComplex*));

        h_H_pos = (long**)malloc(dim*sizeof(long*));
        h_H_vals = (hipDoubleComplex**)malloc(dim*sizeof(hipDoubleComplex*));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation for upper half arrays failed! Error: ";
                cout<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}

	
        long** d_H_pos;
        hipDoubleComplex** d_H_vals;

        status1 = hipMalloc(&d_H_pos, dim*sizeof(long*));
        status2 = hipMalloc(&d_H_vals, dim*sizeof(hipDoubleComplex*));

        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
              cout<<"Memory allocation for device Hamiltonian failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
              return 1;
        }

	/*for(long i = 0; i<dim; i++){
		status1 = hipMalloc(&(h_H_vals[i]), (2*(lattice_Size)+1)*sizeof(hipDoubleComplex));
		status2 = hipMalloc(&(h_H_pos[i]), (2*(lattice_Size)+2)*sizeof(long));
                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                      cout<<"Memory allocation for "<<i<<"th Hamiltonian arrays failed! Error: "<<hipPeekAtLastError()<<endl;
                      return 1;
                }

      }



      status1 = hipMemcpy(d_H_pos, h_H_pos, dim*sizeof(long*), hipMemcpyHostToDevice);
      status2 = hipMemcpy(d_H_vals, h_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyHostToDevice);

      if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
          cout<<"Copy of Hamiltonian array pointers from host to device failed! Error: "<<hipPeekAtLastError()<<endl;
          return 1;
      }

	This code is for the case where we have a device of compute capability <2.0. It's super slow T_T

      */

      CDCarraysalloc<<<(dim/256), 256 >>>(d_H_vals, dim, 2*(lattice_Size) + 1, 0);
      hipDeviceSynchronize();
      longarraysalloc<<<(dim/256), 256>>>(d_H_pos, dim, 2*(lattice_Size) + 2, 0);
      hipDeviceSynchronize();
      
      status1 = hipMemcpy(h_H_vals, d_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyDeviceToHost);
      status2 = hipMemcpy(h_H_pos, d_H_pos, dim*sizeof(long*), hipMemcpyDeviceToHost);    
      
      if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
          cout<<"Copy of Hamiltonian array pointers from host to device failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
          return 1;
      }
     
      SetFirst<<<(dim/256), tpb>>>(d_H_pos, dim, 1); 
      /*	
      for(int jj = 0; jj < dim; jj++){


		status3 = hipMemset((h_H_pos[jj]), 1l, sizeof(long));
                if (status3 != hipSuccess){
                    cout<<"Counting the diagonal element failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }
      } //counting the diagonal element
      */

	// --------------------- Fill up the sparse matrix and compress it to remove extraneous elements ------//
      	double JJ = 1.;

        hipDeviceSynchronize();
	

	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, dim, d_H_vals, d_H_pos, d_Bond, lattice_Size, JJ);

	hipDeviceSynchronize(); //need to make sure all elements are initialized before I start compression
	
	bpg = (dim/256, (((2*lattice_Size + 2)/32) + 1) , 1);
	tpb = (256, 32, 1);
	
	CompressSparse<<<bpg, tpb>>>(d_H_vals, d_H_pos, dim, lattice_Size);
        hipDeviceSynchronize();
	
	long** buffer_H_pos; //created these arrays to hold pointers for me - may be able to remove them later
	hipDoubleComplex** buffer_H_vals;

	status1 = hipHostMalloc(&buffer_H_pos, dim*sizeof(long*));
	status2 = hipHostMalloc(&buffer_H_vals, dim*sizeof(hipDoubleComplex*));

        if ( (status1 != hipSuccess ) || (status2 != hipSuccess ) ){
            cout<<"Memory allocation for Hamiltonian arrays on host failed!"<< hipGetErrorString(hipPeekAtLastError())<<endl;
            return 1;
        }

	//----Copy over the Hamiltonian arrays and sort them-------------//

	for(int ii = 0; ii < dim; ii++){
                long* temp;
                temp = (long*)malloc(sizeof(long));

                status3 = hipMemcpy(temp, &d_H_pos[ii][0], sizeof(long), hipMemcpyDeviceToHost);

                status1 = hipHostMalloc(h_H_pos + ii, (*temp+1)*sizeof(long));
                status2 = hipHostMalloc(h_H_vals + ii, (*temp)*sizeof(hipDoubleComplex));

                if ( (status1 != hipSuccess) ||
                     (status2 != hipSuccess) ||
                     (status3 != hipSuccess) ){
                    
                    cout<<"Memory allocation for "<<ii<<"th host Hamiltonian arrays failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
                    return 1;
                }

		status1 = hipMemcpy(h_H_pos[ii], d_H_pos[ii], (*temp + 1)*sizeof(long), hipMemcpyDeviceToHost);
		status2 = hipMemcpy(h_H_vals[ii], d_H_vals[ii], (*temp)*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Copying "<<ii<<" th Hamiltonian arrays from device to host failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
                    return 1;
                }

		hipFree(&d_H_vals[ii]);
		hipFree(&d_H_pos[ii]);

                hamstruct temphamstruct;

                vector<hamstruct> sortcontainer;

                for (uint jj = 0; jj << h_H_pos[ii][0]; jj++){
                  	temphamstruct.position = h_H_pos[ii][jj+1];
                  	temphamstruct.value = h_H_vals[ii][jj];

                  	sortcontainer.push_back(temphamstruct);

                }

                sort(sortcontainer.begin(), sortcontainer.end());

		for(uint kk = 0; kk < sortcontainer.size(); kk++){
			
			h_H_pos[ii][kk + 1] = (sortcontainer.at(kk)).position;
			h_H_vals[ii][kk] = (sortcontainer.at(kk)).value;
		}


		for(uint ll = 0; ll < (h_H_pos[ii][0] + 1); ll++){
			cout<<"At position: ("<<ii<<", "<<h_H_pos[ii][ll+1]<<": ";
			cout<<h_H_vals[ii][ll].x<<endl;
		}

                free(&temp);
	}

	UpperHalfToFull(h_H_vals, h_H_pos, dim, lattice_Size);

	dim3 tpb2 = ( tpb.x, tpb.x );	
	dim3 bpg2 = ( dim/tpb.x, dim/tpb.x );


	for(long mm = 0; mm < dim; mm++){ //this for loop allocates just enough memory in the device arrays to hold the full row, copies the data, and also finds the true number of nonzero elements
		num_Elem += h_H_pos[mm][0];

                //long* temppos;
                //hipDoubleComplex* tempval;
		
                status1 = hipMalloc(&buffer_H_pos[mm], (h_H_pos[mm][0]+ 1)*sizeof(long));
		status2 = hipMalloc(&buffer_H_vals[mm], (h_H_pos[mm][0])*sizeof(hipDoubleComplex));
		
                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Memory allocation for "<<mm<<"th arrays on device failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
                    return 1;
                }

                status1 = hipMemcpy(buffer_H_pos[mm], h_H_pos[mm], (h_H_pos[mm][0] + 1)*sizeof(long), hipMemcpyHostToHost);
                status2 = hipMemcpy(buffer_H_vals[mm], h_H_vals[mm], (h_H_pos[mm][0])*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
                

                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Array copy from host to buffer failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
                    return 1;
                }

                
	}

        status1 = hipMemcpy(d_H_vals, buffer_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyHostToDevice);
        status2 = hipMemcpy(d_H_pos, buffer_H_pos, dim*sizeof(long*), hipMemcpyHostToDevice);
                                                                                        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
            cout<<"Pointer memory copy from host buffer to device failed!"<< hipGetErrorString( hipPeekAtLastError() )<<endl;     
            return 1;                                                                                       
        }
        
        
	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&hamil_PosRow, num_Elem*sizeof(long));
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(long));

	if ( (status1 != hipSuccess) ||
	     (status2 != hipSuccess) ||
	     (status3 != hipSuccess) ){
		cout<<"Memory allocation for COO representation failed! Error: "<<hipGetErrorString( hipPeekAtLastError() )<<endl;
		return 1;
	}
	
        FullToCOO<<<bpg, tpb>>>(num_Elem, d_H_vals, d_H_pos, hamil_Values, hamil_PosRow, hamil_PosCol, dim); // csr and description initializations happen somewhere else

	for(int nn = 0; nn < dim; nn++){
		hipFree(&d_H_vals[nn]);
		hipFree(&d_H_pos[nn]);
                hipHostFree(&h_H_vals[nn]);
                hipHostFree(&h_H_pos[nn]);
                hipHostFree(&buffer_H_vals[nn]);
                hipHostFree(&buffer_H_pos[nn]);
	}

	hipFree(&d_H_vals); //cleanup
	hipFree(&d_H_pos);

	hipHostFree(&h_H_vals);
	hipHostFree(&h_H_pos);
        hipHostFree(&buffer_H_vals);
        hipHostFree(&buffer_H_pos);
        hipFree(&d_basis);
        hipFree(&d_basis_Position);
        hipFree(&d_Bond);

	return 0;
}

int main(){
        hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024); //have to set a heap size or malloc()s on the device will fail
        
	long* Bond;
	Bond = (long*)malloc(16*3*sizeof(long));
	
	Bond[0] = 0; 	Bond[1] = 1; 	Bond[2] = 2;	Bond[3] = 3;	Bond[4] = 4;	Bond[5] = 5;
	Bond[6] = 6; 	Bond[7] = 7;	Bond[8] = 8;	Bond[9] = 9;	Bond[10] = 10;	Bond[11] = 11;
	Bond[12] = 12;	Bond[13] = 13;	Bond[14] = 14;	Bond[15] = 15;	Bond[16] = 1; 	Bond[17] = 2;
	Bond[18] = 3; 	Bond[19] = 0;	Bond[20] = 5; 	Bond[21] = 6;	Bond[22] = 7;	Bond[23] = 4;
	Bond[24] = 9;	Bond[25] = 10; 	Bond[26] = 11;	Bond[27] = 8;	Bond[28] = 13;	Bond[29] = 14;
	Bond[30] = 15; 	Bond[31] = 12; 	Bond[32] = 4;	Bond[33] = 5;	Bond[34] = 6;	Bond[35] = 7;
	Bond[36] = 8;	Bond[37] = 9;	Bond[38] = 10;	Bond[39] = 11;	Bond[40] = 12;	Bond[41] = 13;
	Bond[42] = 14;	Bond[43] = 15;	Bond[44] = 0;	Bond[45] = 1;	Bond[46] = 2;	Bond[48] = 3;

	hipDoubleComplex* hamil_Values;

	long* hamil_PosRow;

	long* hamil_PosCol;


	int rtn = ConstructSparseMatrix(0, 16, Bond, hamil_Values, hamil_PosRow, hamil_PosCol);

	free(Bond);

	return rtn;
}

__global__ void FillSparse(long* d_basis_Position, long* d_basis, int d_dim, hipDoubleComplex** H_vals, long** H_pos, long* d_Bond, int d_lattice_Size, const double JJ){

	int T0 = blockIdx.y; //my indices!
	int ii = threadIdx.x + blockDim.x*blockIdx.x;

	int si, sj,sk,sl; //spin operators
	unsigned long tempi, tempj, tempod;
	hipDoubleComplex tempD;

	if( ii < d_dim ){
		//Diagonal part----------------
		tempi = d_basis[ii];
		H_pos[ii][1] = d_basis_Position[tempi];

		H_vals[ii][0] = HDiagPart(tempi, d_lattice_Size, d_Bond, JJ);

		//-------------------------------
		//Horizontal bond ---------------
		si = d_Bond[T0];
		tempod = tempi;
		sj = d_Bond[T0 + d_lattice_Size];
	
		tempod ^= (1<<si);   //toggle bit 
		tempod ^= (1<<sj);   //toggle bit 

		if (d_basis_Position[tempod] > ii){ //build only upper half of matrix
        		H_pos[ii][(2*T0)+2] = d_basis_Position[tempod];

        		H_vals[ii][(2*T0)+1] = HOffBondX(T0,tempi, JJ);
			H_pos[ii][0]++; 
      		}

		else {
			H_pos[ii][(2*T0)+2] = -1;
			H_vals[ii][(2*T0)+1] = make_hipDoubleComplex(0., 0.);
		}

		//Vertical bond -----------------
		tempod = tempi;
      		sj = d_Bond[T0 + 2*d_lattice_Size];
      		tempod ^= (1<<si);   //toggle bit 
     		tempod ^= (1<<sj);   //toggle bit 
      		if (d_basis_Position[tempod] > ii){ 
        		H_pos[ii][(2*T0)+3] = d_basis_Position[tempod];
   
        		H_vals[ii][(2*T0)+2] = HOffBondY(T0,tempi, JJ);
			H_pos[ii][0]++;
      		}

		else {
			H_pos[ii][(2*T0)+3] = -1;
			H_vals[ii][(2*T0)+2] = make_hipDoubleComplex(0., 0.);
		}
	}
}

/* Function: CompressSparse - this function takes the sparse matrix with lots of "buffer" memory sitting on the end of each array, and compresses it down to get rid of the extra memory
Inputs:	H_vals - an array of arrays of Hamiltonian values
	H_pos - an array of arrays of value positions in columns
	d_dim - the dimension of the Hamiltonian, stored on the device
	lattice_Size - the number of lattice sites
Outputs: H_vals - an array of smaller arrays than before
	 H_pos - see above

*/
__global__ void CompressSparse(hipDoubleComplex** H_vals, long** H_pos, int d_dim, const int lattice_Size){

	int row = blockDim.x*blockIdx.x + threadIdx.x;
	int col = blockDim.y*blockIdx.y + threadIdx.y;

	int iter = 0;

	if (row < d_dim){

		// the basic idea here is to have each x thread go to the ith row, and each y thread go to the jth element of that row. then using a set of __shared__ temp arrays, we read in the Hamiltonian values and do our comparisons n stuff on them
		__shared__ int count;
		count = H_pos[row][0];

		const int size1 = 2*lattice_Size + 2;
		const int size2 = 2*lattice_Size + 1;
	
		__shared__ long s_H_pos[34]; //hardcoded for now because c++ sucks
		__shared__ hipDoubleComplex s_H_vals[33];
		

		if (col < size2){
			s_H_pos[col] = H_pos[row][col];
			s_H_vals[col] = H_vals[row][col];
		}

		if (col == size2){
			s_H_vals[col] = H_vals[row][col];
		} //loading the Hamiltonian information into shared memory

		__syncthreads(); // have to make sure all loading is done before we start anything else

		long* temp_pos = (long*)malloc((count + 1)*sizeof(long));
		hipDoubleComplex* temp_val = (hipDoubleComplex*)malloc((count)*sizeof(hipDoubleComplex));
                /*
                if ( temp_pos == (long*)NULL){
                  printf("Allocation of temp_pos in iteration %d failed! \n", i);
                }	

                if ( temp_val == (hipDoubleComplex*)NULL){
                  printf("Allocation of temp_val in iteration %d failed! \n", i);
                }
		*/
		temp_pos[0] = count;

		for(int j = 0; j < size2; j++){
			if( (s_H_pos[j+1] != -1) ){
				temp_pos[iter+1] = s_H_pos[j+1];
				temp_val[iter] = s_H_vals[j];
				iter++;
			}
		 
		}

		free(H_pos[row]); //switching out the old ones for the new
		free(H_vals[row]);
		
		H_pos[row] = temp_pos;
		H_vals[row] = temp_val;

		
	}
}

//this function takes the upper half form I had from FillSparse and CompressSparse and fills out the bottom half of the matrix - since there are so many comparisons it's probably faster to just do this on CPU
__host__ void UpperHalfToFull(hipDoubleComplex** H_vals, long** H_pos, long dim, int lattice_Size) {

	for(int ii = 0; ii<dim; ii++){

		long size = dim - H_pos[ii][0];
		hipDoubleComplex* temp;
		long* temp_col;

		temp = (hipDoubleComplex*)malloc(size*sizeof(hipDoubleComplex));
		temp_col = (long*)malloc(size*sizeof(long));

		long iter = 0;

		for(int jj = 0; jj<ii; jj++){
			for(int kk = 1; kk <= H_pos[jj][0]; kk++){

				if(H_pos[jj][kk] = ii){
					
					temp[iter] = H_vals[jj][kk-1];
					temp_col[iter] = jj;
					iter++;
				}

			}
		}


		hipDoubleComplex temp_vals[H_pos[ii][0] + iter];
		long temp_pos[H_pos[ii][0] + iter + 1];
		temp_pos[0] = iter + H_pos[ii][0]; //we'll need this number later!
		
		for(int ll = 0; ll < H_pos[ii][0] + iter; ll++){
			if (ll < iter){
				temp_vals[ll] = temp[ll];
				temp_pos[ll+1] = temp_col[ll];

			}
			else{
				temp_vals[ll] = H_vals[ii][ll-iter];
				temp_pos[ll+1] = H_pos[ii][ll-iter + 1];
			}
		}
		
		free(&H_vals[ii]);
		free(&H_pos[ii]);
		
		H_vals[ii] = temp_vals;
		H_pos[ii] = temp_pos;
	}

} 

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
	 H_vals - the Hamiltonian values
	 H_pos - the Hamiltonian positions
	 hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(long num_Elem, hipDoubleComplex** H_vals, long** H_pos, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol, long dim){

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.x*blockIdx.y;

	const int size = H_pos[i][0];

	long start = 0;

	__shared__ long s_H_pos[34]; //hardcoded for now because c++ sucks
	__shared__ hipDoubleComplex s_H_vals[33];

	if (j < size){
			s_H_pos[j] = H_pos[i][j];
			s_H_vals[j] = H_vals[i][j];
	}

	if (j == size){
			s_H_vals[j] = H_vals[i][j];
	} //loading the Hamiltonian information into shared memory


	if (i < size){
		for(int k = 0; k < i; k++){
			start += H_pos[k][0];
		} //need to know where to start sticking values into the COO arrays


		if(j < size){
			hamil_Values[start + j] = s_H_vals[j];
			hamil_PosRow[start + j] = i;
			hamil_PosCol[start + j] = s_H_pos[j+1];
		}
	}
}

