#include "hip/hip_runtime.h"
#include<hamiltonian.h>



__host__ void GetBasis(int dim, int Nsite, int Sz, long* basis_Position, long* basis){
	unsigned long temp = 0;
	
	for (unsigned long i1=0; i1<dim; i1++){
		basis[il] = -1;
	}
	
	for (unsigned long i1=0; i1<dim; i1++){
      		temp = 0;
      		for (int sp =0; sp<Nsite; sp++)
          		temp += (i1>>sp)&1;  //unpack bra
      		if (temp==(Nsite/2+Sz) ){ 
          		basis[i1] = il;
          		basis_Position[il] = il -1;
      		}
  	} 

}

__device__ double HOffBondX(const int si, const long bra, const double JJ){

	double valH;
  	int S0, S1;
  	int T0, T1;

  	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

  	return valH;


} 

__device__ double HOffBondY(const int si, const long bra, const double JJ){

	double valH;
  	int S0, S1;
  	int T0, T1;

  	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

  	return valH;


} 

int ConstructSparseMatrix(enum model_Type, int lattice_Size, long* Bond){
	
	unsigned long num_Elem; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	switch (model_Type){
		case 0: num_Elem = 219648;
		case 1: num_Elem = 10; //guesses
	}

	hipDoubleComplex* hamil_Values;
	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));

	long* hamil_PosRow;
	status2 = hipMalloc(&hamil_PosRow, (1<<lattice_Size)*sizeof(long));

	long* hamil_PosCol;
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(long));

	if ( (status1 != hipSuccess) ||
	     (status2 != hipSuccess) ||
	     (status3 != hipSuccess) ){
		printf("Memory allocation for COO representation failed!");
		return 1;
	}		

	long* d_Bond;
	status1 = hipMalloc(&d_Bond, sizeof(Bond)*sizeof(long));

	status2 = hipMemcpy(d_Bond, Bond, sizeof(Bond)*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		printf("Memory allocation and copy for bond data failed!");
		return 1;
	}
		
	for (int ch=1; ch<Nsite; ch++) dim *=2;

	long basis_Position[dim];
	long basis[dim];
	
	GetBasis(dim, lattice_Size, Sz, &basis_Position, &basis);

	long* d_basis_Position;
	long* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(long));
	status2 = hipMalloc(&d_basis, dim*sizeof(long));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		printf("Memory allocation for basis arrays failed!");
		return 1;
	}

	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(long), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, dim*sizeof(long), cudaMemcopyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		printf("Memory copy for basis arrays failed!");
		return 1;
	}

	dim3 bpg = (ceil(dim/1024.), lattice_Size);
	int tpb = 1024; //these are going to need to depend on dim and Nsize

	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, d_dim, H_vals, H_pos, d_Bond); //each block 

}


__global__ void FillSparse(long* d_basis_Position, long* d_basis, int* d_dim, hipDoubleComplex** H_vals, long** H_pos, long* d_Bond);

	int T0 = blockIdx.y; //my indices!
	int ii = threadIdx.x + 1024*blockIdx.x;

	int si, sj,sk,sl; //spin operators
	unsigned long tempi, tempj, tempod;
	double tempD;

	tempi = d_basis[ii];
	H_pos[ii][0] = d_basis_Position[tempi];

	tempD = HDiagPart(tempi);

	H_vals[ii][0] = tempD;

	si = d_Bond[T0];
	tempod = tempi;
	sj = d_Bond[T0 + lattice_Size];
	
	tempod ^= (1<<si);   //toggle bit 
	tempod ^= (1<<sj);   //toggle bit 

	if (d_basis_Position[tempod] != -1 && d_basis_Position[tempod] > ii){ //build only upper half of matrix
        	H_pos[ii][T0] = d_basis_Position[tempod];
        	tempD = HOFFdBondX(T0,tempi);
        	H_vals[ii][T0] = tempD; 
      	}
