#include "hip/hip_runtime.h"
#include"hamiltonian.h"


/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
	lattice_Size - the number of sites
	Sz - the value of the Sz operator
	basis_Position[] - an empty array that records the positions of the basis
	basis - an empty array that records the basis
Outputs:	basis_Position - a full array now
		basis[] - a full array now

*/
__host__ void GetBasis(int dim, int lattice_Size, int Sz, long basis_Position[], long basis[]){
	unsigned long temp = 0;

	for (unsigned long i1=0; i1<dim; i1++){
      		temp = 0;
		basis_Position[i1] = -1;
      		for (int sp =0; sp<lattice_Size; sp++){
          		temp += (i1>>sp)&1;
		}  //unpack bra
      		if (temp==(lattice_Size/2+Sz) ){ 
          		basis[i1] = i1;
          		basis_Position[i1] = i1 -1;
      		}
  	} 

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
        bra - the state
        JJ - the coupling constant
Outputs:  valH - the value of the Hamiltonian 

*/

__device__ hipDoubleComplex HOffBondX(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0.); //contribution from the J part of the Hamiltonian

  	return valH;


} 

__device__ hipDoubleComplex HOffBondY(const int si, const long bra, const double JJ){

	hipDoubleComplex valH;
  	int S0, S1;
  	int T0, T1;

  	valH = make_hipDoubleComplex( JJ*0.5 , 0. ); //contribution from the J part of the Hamiltonian

  	return valH;


}

__device__ hipDoubleComplex HDiagPart(const long bra, int lattice_Size, long* d_Bond, const double JJ){

  int S0b,S1b ;  //spins (bra 
  int T0,T1;  //site
  int P0, P1, P2, P3; //sites for plaquette (Q)
  int s0p, s1p, s2p, s3p;
  hipDoubleComplex valH = make_hipDoubleComplex( 0. , 0.);

  for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

    T0 = d_Bond[Ti]; //lower left spin
    S0b = (bra>>T0)&1;  
    //if (T0 != Ti) cout<<"Square error 3\n";
    T1 = d_Bond[Ti + lattice_Size]; //first bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);
    T1 = d_Bond[Ti + 2*lattice_Size]; //second bond
    S1b = (bra>>T1)&1;  //unpack bra
    valH.x += JJ*(S0b-0.5)*(S1b-0.5);

  }//T0

  //cout<<bra<<" "<<valH<<endl;

  return valH;

}//HdiagPart 



/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
	lattice_Size - the number of lattice sites
	Bond - the bond values ??
	hamil_Values - an empty pointer for a device array containing the values 
	hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
	hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs:  hamil_Values - a pointer to a device array containing the values 
	hamil_PosRow - a pointer to a device array containing the locations of each value in a row
	hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


int ConstructSparseMatrix(int model_Type, int lattice_Size, long* Bond, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol){
	
	unsigned long num_Elem; // the total number of elements in the matrix, will get this (or an estimate) from the input types
	hipError_t status1, status2, status3;

	switch (model_Type){
		case 0: num_Elem = 219648;
		case 1: num_Elem = 10; //guesses
	}

	int dim = 2;

	
	long* d_Bond;
	status1 = hipMalloc(&d_Bond, 3*lattice_Size*sizeof(long));

	status2 = hipMemcpy(d_Bond, Bond, 3*lattice_Size*sizeof(long), hipMemcpyHostToDevice);

	status1 = hipPeekAtLastError();

	/*if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation and copy for bond data failed!";
		return 1;
	}*/

	if (status1 != hipSuccess){
		cout<<"Bond data allocation error: "<<status1<<endl;
		return 1;
	}

	if (status2 != hipSuccess){
		cout<<"Bond data copy error: "<<status2<<endl;
		return 1;
	}


	for (int ch=1; ch<lattice_Size; ch++) dim *=2;

        cout<<dim<<endl;

	long basis_Position[dim];
	long basis[dim];

	int Sz = 0;
	
	GetBasis(dim, lattice_Size, Sz, basis_Position, basis);

	long* d_basis_Position;
	long* d_basis;

	status1 = hipMalloc(&d_basis_Position, dim*sizeof(long));
	status2 = hipMalloc(&d_basis, dim*sizeof(long));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation for basis arrays failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

	status1 = hipMemcpy(d_basis_Position, basis_Position, dim*sizeof(long), hipMemcpyHostToDevice);
	status2 = hipMemcpy(d_basis, basis, dim*sizeof(long), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory copy for basis arrays failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

	dim3 bpg = (dim/256, lattice_Size);
	int tpb = 256; //these are going to need to depend on dim and Nsize

	long** h_H_pos;
	hipDoubleComplex** h_H_vals; 

	//status1 = hipMalloc(&H_pos, dim*sizeof(long*));
	//status2 = hipMalloc(&H_vals, dim*sizeof(hipDoubleComplex*));

        h_H_pos = (long**)malloc(dim*sizeof(long*));
        h_H_vals = (hipDoubleComplex**)malloc(dim*sizeof(hipDoubleComplex*));

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation for upper half arrays failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

	int* d_dim;
	status1 = hipMalloc(&d_dim, sizeof(int));
	status2 = hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation and copy for dimension failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

	int* d_lattice_Size;
	status1 = hipMalloc(&d_lattice_Size, sizeof(int));
	status2 = hipMemcpy(d_lattice_Size, &lattice_Size, sizeof(int), hipMemcpyHostToDevice);

	if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
		cout<<"Memory allocation and copy for lattice size failed! Error: ";
                cout<<hipPeekAtLastError()<<endl;
		return 1;
	}

        long** d_H_pos;
        hipDoubleComplex** d_H_vals;

        status1 = hipMalloc(&d_H_pos, dim*sizeof(long*));
        status2 = hipMalloc(&d_H_vals, dim*sizeof(hipDoubleComplex*));

        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
              cout<<"Memory allocation for device Hamiltonian failed! Error: "<<hipPeekAtLastError()<<endl;
              return 1;
        }

	/*for(long i = 0; i<dim; i++){
		status1 = hipMalloc(&(h_H_vals[i]), (2*(lattice_Size)+1)*sizeof(hipDoubleComplex));
		status2 = hipMalloc(&(h_H_pos[i]), (2*(lattice_Size)+2)*sizeof(long));
                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                      cout<<"Memory allocation for "<<i<<"th Hamiltonian arrays failed! Error: "<<hipPeekAtLastError()<<endl;
                      return 1;
                }

      }



      status1 = hipMemcpy(d_H_pos, h_H_pos, dim*sizeof(long*), hipMemcpyHostToDevice);
      status2 = hipMemcpy(d_H_vals, h_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyHostToDevice);

      if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
          cout<<"Copy of Hamiltonian array pointers from host to device failed! Error: "<<hipPeekAtLastError()<<endl;
          return 1;
      }

      */

      CDCarraysalloc<<<(dim/256), 256 >>>(d_H_vals, dim, 2*(lattice_Size) + 1, 0);
      hipDeviceSynchronize();
      longarraysalloc<<<(dim/256), 256>>>(d_H_pos, dim, 2*(lattice_Size) + 2, 0);
      
      
      status1 = hipMemcpy(h_H_vals, d_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyDeviceToHost);
      status2 = hipMemcpy(h_H_pos, d_H_pos, dim*sizeof(long*), hipMemcpyDeviceToHost);    
      
      if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
          cout<<"Copy of Hamiltonian array pointers from host to device failed! Erro: "<<hipPeekAtLastError()<<endl;
          return 1;
      }
      
      	
      for(int jj = 0; jj < dim; jj++){
		status3 = hipMemset(&h_H_pos[jj][0], 1, sizeof(int));
                if (status3 != hipSuccess){
                    cout<<"Counting the diagonal element failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }
      } //counting the diagonal element

      double JJ = 1.;

	FillSparse<<<bpg, tpb>>>(d_basis_Position, d_basis, *d_dim, d_H_vals, d_H_pos, d_Bond, *d_lattice_Size, JJ);

	hipDeviceSynchronize(); //need to make sure all elements are initialized before I start compression
	
	CompressSparse<<<bpg, tpb>>>(d_H_vals, d_H_pos, *d_dim, *d_lattice_Size);

	long** buffer_H_pos;
	hipDoubleComplex** buffer_H_vals;

	status1 = hipHostMalloc(&buffer_H_pos, dim*sizeof(long*));
	status2 = hipHostMalloc(&buffer_H_vals, dim*sizeof(hipDoubleComplex*));

        if ( (status1 != hipSuccess ) || (status2 != hipSuccess ) ){
            cout<<"Memory allocation for Hamiltonian arrays on host failed!"<<endl;
            return 1;
        }

	for(int ii = 0; ii < dim; ii++){
                long* temp;
                temp = (long*)malloc(sizeof(long));

                status3 = hipMemcpy(temp, &d_H_pos[ii][0], sizeof(long), hipMemcpyDeviceToHost);

                status1 = hipHostMalloc(h_H_pos + ii, (*temp+1)*sizeof(long));
                status2 = hipHostMalloc(h_H_vals + ii, (*temp)*sizeof(hipDoubleComplex));

                if ( (status1 != hipSuccess) ||
                     (status2 != hipSuccess) ||
                     (status3 != hipSuccess) ){
                    
                    cout<<"Memory allocation for "<<ii<<"th host Hamiltonian arrays failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }

		status1 = hipMemcpy(h_H_pos[ii], d_H_pos[ii], (*temp + 1)*sizeof(long), hipMemcpyDeviceToHost);
		status2 = hipMemcpy(h_H_vals[ii], d_H_vals[ii], (*temp)*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Copying "<<ii<<" th Hamiltonian arrays from device to host failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }

		hipFree(&d_H_vals[ii]);
		hipFree(&d_H_pos[ii]);

                hamstruct temphamstruct;

                vector<hamstruct> sortcontainer;

                for (uint jj = 0; jj << h_H_pos[ii][0]; jj++){
                  	temphamstruct.position = h_H_pos[ii][jj+1];
                  	temphamstruct.value = h_H_vals[ii][jj];

                  	sortcontainer.push_back(temphamstruct);

                }

                sort(sortcontainer.begin(), sortcontainer.end());

		for(uint kk = 0; kk < sortcontainer.size(); kk++){
			
			h_H_pos[ii][kk + 1] = (sortcontainer.at(kk)).position;
			h_H_vals[ii][kk] = (sortcontainer.at(kk)).value;
		}


		for(uint ll = 0; ll < (h_H_pos[ii][0] + 1); ll++){
			cout<<"At position: ("<<ii<<", "<<h_H_pos[ii][ll+1]<<": ";
			cout<<h_H_vals[ii][ll].x<<endl;
		}

                free(&temp);
	}

	UpperHalfToFull(h_H_vals, h_H_pos, dim, lattice_Size);

	dim3 tpb2 = ( tpb, tpb );	
	dim3 bpg2 = ( dim/tpb, dim/tpb );

	num_Elem = 0;

	for(long mm = 0; mm < dim; mm++){ //this for loop allocates just enough memory in the device arrays to hold the full row, copies the data, and also finds the true number of nonzero elements
		num_Elem += h_H_pos[mm][0];

                //long* temppos;
                //hipDoubleComplex* tempval;
		
                status1 = hipMalloc(&buffer_H_pos[mm], (h_H_pos[mm][0]+ 1)*sizeof(long));
		status2 = hipMalloc(&buffer_H_vals[mm], (h_H_pos[mm][0])*sizeof(hipDoubleComplex));
		
                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Memory allocation for "<<mm<<"th arrays on device failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }

                status1 = hipMemcpy(buffer_H_pos[mm], h_H_pos[mm], (h_H_pos[mm][0] + 1)*sizeof(long), hipMemcpyHostToHost);
                status2 = hipMemcpy(buffer_H_vals[mm], h_H_vals[mm], (h_H_pos[mm][0])*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
                

                if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
                    cout<<"Array copy from host to buffer failed! Error: "<<hipPeekAtLastError()<<endl;
                    return 1;
                }

                
	}

        status1 = hipMemcpy(d_H_vals, buffer_H_vals, dim*sizeof(hipDoubleComplex*), hipMemcpyHostToDevice);
        status2 = hipMemcpy(d_H_pos, buffer_H_pos, dim*sizeof(long*), hipMemcpyHostToDevice);
                                                                                        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
            cout<<"Pointer memory copy from host buffer to device failed!"<<endl;     
            return 1;                                                                                       
        }
        
        long* d_num_Elem;
        status1 = hipMalloc(&d_num_Elem, sizeof(long));
        status2 = hipMemcpy(d_num_Elem, &num_Elem, sizeof(long), hipMemcpyHostToDevice);

        if ( (status1 != hipSuccess) || (status2 != hipSuccess) ){
            cout<<"Memory allocation and copy for number of elements failed! Error: "<<hipPeekAtLastError()<<endl;
            return 1;
        }


	status1 = hipMalloc(&hamil_Values, num_Elem*sizeof(hipDoubleComplex));
	status2 = hipMalloc(&hamil_PosRow, num_Elem*sizeof(long));
	status3 = hipMalloc(&hamil_PosCol, num_Elem*sizeof(long));

	if ( (status1 != hipSuccess) ||
	     (status2 != hipSuccess) ||
	     (status3 != hipSuccess) ){
		cout<<"Memory allocation for COO representation failed! Error: "<<hipPeekAtLastError()<<endl;
		return 1;
	}
	
        FullToCOO<<<bpg2, tpb2>>>(*d_num_Elem, d_H_vals, d_H_pos, hamil_Values, hamil_PosRow, hamil_PosCol); // csr and description initializations happen somewhere else

	for(int nn = 0; nn < dim; nn++){
		hipFree(&d_H_vals[nn]);
		hipFree(&d_H_pos[nn]);
                hipHostFree(&h_H_vals[nn]);
                hipHostFree(&h_H_pos[nn]);
                hipHostFree(&buffer_H_vals[nn]);
                hipHostFree(&buffer_H_pos[nn]);
	}

	hipFree(&d_H_vals); //cleanup
	hipFree(&d_H_pos);
	hipFree(&d_dim);
        hipFree(&d_lattice_Size);
        hipFree(&d_num_Elem);
	hipHostFree(&h_H_vals);
	hipHostFree(&h_H_pos);
        hipHostFree(&buffer_H_vals);
        hipHostFree(&buffer_H_pos);
        hipFree(&d_basis);
        hipFree(&d_basis_Position);
        hipFree(&d_Bond);

	return 0;
}

int main(){

	long* Bond;
	Bond = (long*)malloc(16*3*sizeof(long));
	
	Bond[0] = 0; 	Bond[1] = 1; 	Bond[2] = 2;	Bond[3] = 3;	Bond[4] = 4;	Bond[5] = 5;
	Bond[6] = 6; 	Bond[7] = 7;	Bond[8] = 8;	Bond[9] = 9;	Bond[10] = 10;	Bond[11] = 11;
	Bond[12] = 12;	Bond[13] = 13;	Bond[14] = 14;	Bond[15] = 15;	Bond[16] = 1; 	Bond[17] = 2;
	Bond[18] = 3; 	Bond[19] = 0;	Bond[20] = 5; 	Bond[21] = 6;	Bond[22] = 7;	Bond[23] = 4;
	Bond[24] = 9;	Bond[25] = 10; 	Bond[26] = 11;	Bond[27] = 8;	Bond[28] = 13;	Bond[29] = 14;
	Bond[30] = 15; 	Bond[31] = 12; 	Bond[32] = 4;	Bond[33] = 5;	Bond[34] = 6;	Bond[35] = 7;
	Bond[36] = 8;	Bond[37] = 9;	Bond[38] = 10;	Bond[39] = 11;	Bond[40] = 12;	Bond[41] = 13;
	Bond[42] = 14;	Bond[43] = 15;	Bond[44] = 0;	Bond[45] = 1;	Bond[46] = 2;	Bond[48] = 3;

	hipDoubleComplex* hamil_Values;

	long* hamil_PosRow;

	long* hamil_PosCol;


	int rtn = ConstructSparseMatrix(0, 16, Bond, hamil_Values, hamil_PosRow, hamil_PosCol);

	free(Bond);

	return rtn;
}

__global__ void FillSparse(long* d_basis_Position, long* d_basis, int d_dim, hipDoubleComplex** H_vals, long** H_pos, long* d_Bond, int d_lattice_Size, const double JJ){

	int T0 = blockIdx.y; //my indices!
	int ii = threadIdx.x + 256*blockIdx.x;

	int si, sj,sk,sl; //spin operators
	unsigned long tempi, tempj, tempod;
	hipDoubleComplex tempD;

	if( ii < d_dim ){
		//Diagonal part----------------
		tempi = d_basis[ii];
		H_pos[ii][1] = d_basis_Position[tempi];

		H_vals[ii][0] = HDiagPart(tempi, d_lattice_Size, d_Bond, JJ);

		//-------------------------------
		//Horizontal bond ---------------
		si = d_Bond[T0];
		tempod = tempi;
		sj = d_Bond[T0 + d_lattice_Size];
	
		tempod ^= (1<<si);   //toggle bit 
		tempod ^= (1<<sj);   //toggle bit 

		if (d_basis_Position[tempod] > ii){ //build only upper half of matrix
        		H_pos[ii][(2*T0)+2] = d_basis_Position[tempod];

        		H_vals[ii][(2*T0)+1] = HOffBondX(T0,tempi, JJ);
			H_pos[ii][0]++; 
      		}

		else {
			H_pos[ii][(2*T0)+2] = -1;
			H_vals[ii][(2*T0)+1] = make_hipDoubleComplex(0., 0.);
		}

		//Vertical bond -----------------
		tempod = tempi;
      		sj = d_Bond[T0 + 2*d_lattice_Size];
      		tempod ^= (1<<si);   //toggle bit 
     		tempod ^= (1<<sj);   //toggle bit 
      		if (d_basis_Position[tempod] > ii){ 
        		H_pos[ii][(2*T0)+3] = d_basis_Position[tempod];
   
        		H_vals[ii][(2*T0)+2] = HOffBondY(T0,tempi, JJ);
			H_pos[ii][0]++;
      		}

		else {
			H_pos[ii][(2*T0)+3] = -1;
			H_vals[ii][(2*T0)+2] = make_hipDoubleComplex(0., 0.);
		}
	}
}

/* Function: CompressSparse - this function takes the sparse matrix with lots of "buffer" memory sitting on the end of each array, and compresses it down to get rid of the extra memory
Inputs:	H_vals - an array of arrays of Hamiltonian values
	H_pos - an array of arrays of value positions in columns
	d_dim - the dimension of the Hamiltonian, stored on the device
	lattice_Size - the number of lattice sites
Outputs: H_vals - an array of smaller arrays than before
	 H_pos - see above

*/
__global__ void CompressSparse(hipDoubleComplex** H_vals, long** H_pos, int d_dim, int lattice_Size){

	int i = 256*blockIdx.x + threadIdx.x;

	int iter = 0;

	if (i < d_dim){
	
		long* temp_pos = (long*)malloc(H_pos[i][0]*sizeof(long));
		hipDoubleComplex* temp_val = (hipDoubleComplex*)malloc((H_pos[i][0]+ 1)*sizeof(long));	

		temp_pos[0] = H_pos[i][0];

		for(int j = 0; j < 2*lattice_Size + 1; j++){
			if( (H_pos[i][j+1] != -1) ){
				temp_pos[iter+1] = H_pos[i][j+1];
				temp_val[iter] = H_vals[i][j];
				iter++;
			}
		 
		}

		free(H_pos[i]); //switching out the old ones for the new
		free(H_vals[i]);
		
		H_pos[i] = temp_pos;
		H_vals[i] = temp_val;

		
	}
}

//this function takes the upper half form I had from FillSparse and CompressSparse and fills out the bottom half of the matrix - since there are so many comparisons it's probably faster to just do this on CPU
__host__ void UpperHalfToFull(hipDoubleComplex** H_vals, long** H_pos, long dim, int lattice_Size) {

	for(int ii = 0; ii<dim; ii++){

		long size = dim - H_pos[ii][0];
		hipDoubleComplex* temp;
		long* temp_col;

		temp = (hipDoubleComplex*)malloc(size*sizeof(hipDoubleComplex));
		temp_col = (long*)malloc(size*sizeof(long));

		long iter = 0;

		for(int jj = 0; jj<ii; jj++){
			for(int kk = 1; kk <= H_pos[jj][0]; kk++){

				if(H_pos[jj][kk] = ii){
					
					temp[iter] = H_vals[jj][kk-1];
					temp_col[iter] = jj;
					iter++;
				}

			}
		}


		hipDoubleComplex temp_vals[H_pos[ii][0] + iter];
		long temp_pos[H_pos[ii][0] + iter + 1];
		temp_pos[0] = iter + H_pos[ii][0]; //we'll need this number later!
		
		for(int ll = 0; ll < H_pos[ii][0] + iter; ll++){
			if (ll < iter){
				temp_vals[ll] = temp[ll];
				temp_pos[ll+1] = temp_col[ll];

			}
			else{
				temp_vals[ll] = H_vals[ii][ll-iter];
				temp_pos[ll+1] = H_pos[ii][ll-iter + 1];
			}
		}
		
		free(&H_vals[ii]);
		free(&H_pos[ii]);
		
		H_vals[ii] = temp_vals;
		H_pos[ii] = temp_pos;
	}

} 

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
	 H_vals - the Hamiltonian values
	 H_pos - the Hamiltonian positions
	 hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(long num_Elem, hipDoubleComplex** H_vals, long** H_pos, hipDoubleComplex* hamil_Values, long* hamil_PosRow, long* hamil_PosCol){

	int i = threadIdx.x + 256*blockIdx.x;
	int j = threadIdx.y + 256*blockIdx.y;

	long start = 0;

	if (i < sizeof(H_vals)){
		for(int k = 0; k < i; k++){
			start += H_pos[k][0];
		} //need to know where to start sticking values into the COO arrays


		if(j < H_pos[i][0]){
			hamil_Values[start + j] = H_vals[i][j];
			hamil_PosRow[start + j] = i;
			hamil_PosCol[start + j] = H_pos[i][j+1];
		}
	}
}

