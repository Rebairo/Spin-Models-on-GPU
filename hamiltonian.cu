#include "hip/hip_runtime.h"
#include"hamiltonian.h"

/* NOTE: this function uses FORTRAN style matrices, where the values and positions are stored in a ONE dimensional array! Don't forget this! */


int main(){


	int** Bond;

	int how_many = 30;
  	Bond = (int**)malloc(how_many*sizeof(int*));
	d_hamiltonian* hamil_lancz = (d_hamiltonian*)malloc(how_many*sizeof(d_hamiltonian));
	int* nsite = (int*)malloc(how_many*sizeof(int));
	int* Sz = (int*)malloc(how_many*sizeof(int));
	float* JJ = (float*)malloc(how_many*sizeof(float));	
	int* model_type = (int*)malloc(how_many*sizeof(int));

	for(int i = 0; i < how_many; i++){
		Bond[i] = (int*)malloc(16*3*sizeof(int));
  		Bond[i][0] = 0; Bond[i][1] = 1; Bond[i][2] = 2; Bond[i][3] = 3; Bond[i][4] = 4;
  		Bond[i][5] = 5; Bond[i][6] = 6; Bond[i][7] = 7; Bond[i][8] = 8; Bond[i][9] = 9;
  		Bond[i][10] = 10; Bond[i][11] = 11; Bond[i][12] = 12; Bond[i][13] = 13; Bond[i][14] = 14;
  		Bond[i][15] = 15; Bond[i][16] = 1; Bond[i][17] = 2; Bond[i][18] = 3; Bond[i][19] = 0;
  		Bond[i][20] = 5; Bond[i][21] = 6; Bond[i][22] = 7; Bond[i][23] = 4; Bond[i][24] = 9;
  		Bond[i][25] = 10; Bond[i][26] = 11; Bond[i][27] = 8; Bond[i][28] = 13; Bond[i][29] = 14;
  		Bond[i][30] = 15; Bond[i][31] = 12; Bond[i][32] = 4; Bond[i][33] = 5; Bond[i][34] = 6;
  		Bond[i][35] = 7; Bond[i][36] = 8; Bond[i][37] = 9; Bond[i][38] = 10; Bond[i][39] = 11;
  		Bond[i][40] = 12; Bond[i][41] = 13; Bond[i][42] = 14; Bond[i][43] = 15; Bond[i][44] = 0;
  		Bond[i][45] = 1; Bond[i][46] = 2; Bond[i][47] = 3;

		nsite[i] = 16;
		Sz[i] = 0;
		JJ[i] = 1.f;
		model_type[i] = 0;
	}
	

  	int dim;

	int* num_Elem = ConstructSparseMatrix(how_many, model_type, nsite, Bond, hamil_lancz, JJ, Sz );

	return 0;
}

__host__ __device__ int idx(int i, int j, int lda){
  
  return (j + (i*lda));
}


/* Function GetBasis - fills two arrays with information about the basis
Inputs: dim - the initial dimension of the Hamiltonian
lattice_Size - the number of sites
Sz - the value of the Sz operator
basis_Position[] - an empty array that records the positions of the basis
basis - an empty array that records the basis
Outputs: basis_Position - a full array now
basis[] - a full array now

*/
__host__ int GetBasis(int dim, int lattice_Size, int Sz, int basis_Position[], int basis[]){
unsigned int temp = 0;
	int realdim = 0;

	for (unsigned int i1=0; i1<dim; i1++){
		temp = 0;
		basis_Position[i1] = -1;
		for (int sp =0; sp<lattice_Size; sp++){
			temp += (i1>>sp)&1;
		} //unpack bra
		//if (temp==(lattice_Size/2+Sz) ){
			basis[realdim] = i1;
			basis_Position[i1] = realdim;
			realdim++;
			//cout<<basis[realdim]<<" "<<basis_Position[i1]<<endl;
		//}
}

return realdim;

}

/* Function HOffBondX
Inputs: si - the spin operator in the x direction
bra - the state
JJ - the coupling constant
Outputs: valH - the value of the Hamiltonian

*/

__device__ float HOffBondX(const int si, const int bra, const float JJ){

	float valH;
	//int S0, S1;
	//int T0, T1;

	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

	return valH;

}

__device__ float HOffBondY(const int si, const int bra, const float JJ){

	float valH;
	//int S0, S1;
	//int T0, T1;

	valH = JJ*0.5; //contribution from the J part of the Hamiltonian

	return valH;


}

__device__ float HDiagPart(const int bra, int lattice_Size, int3* d_Bond, const float JJ){

	int S0b,S1b ; //spins (bra
	int T0,T1; //site
	//int P0, P1, P2, P3; //sites for plaquette (Q)
	//int s0p, s1p, s2p, s3p;
	float valH = 0.f;

	for (int Ti=0; Ti<lattice_Size; Ti++){
    //***HEISENBERG PART

		T0 = (d_Bond[Ti]).x; //lower left spin
		S0b = (bra>>T0)&1;
		//if (T0 != Ti) cout<<"Square error 3\n";
		T1 = (d_Bond[Ti]).y; //first bond
		S1b = (bra>>T1)&1; //unpack bra
		valH += JJ*(S0b-0.5)*(S1b-0.5);
		T1 = (d_Bond[Ti]).z; //second bond
		S1b = (bra>>T1)&1; //unpack bra
		valH += JJ*(S0b-0.5)*(S1b-0.5);

	}//T0

	//cout<<bra<<" "<<valH<<endl;

	return valH;

}//HdiagPart

/* Function: ConstructSparseMatrix:

Inputs: model_Type - tells this function how many elements there could be, what generating functions to use, etc. Presently only supports Heisenberg
lattice_Size - the number of lattice sites
Bond - the bond values ??
hamil_Values - an empty pointer for a device array containing the values
hamil_PosRow - an empty pointer for a device array containing the locations of each value in a row
hamil_PosCol - an empty pointer to a device array containing the locations of each values in a column

Outputs: hamil_Values - a pointer to a device array containing the values
hamil_PosRow - a pointer to a device array containing the locations of each value in a row
hamil_PosCol - a pointer to a device array containing the locations of each values in a column

*/


__host__ int* ConstructSparseMatrix(const int how_many, int* model_Type, int* lattice_Size, int** Bond, d_hamiltonian*& hamil_lancz, float* JJ, int* Sz ){


	//hipSetDevice(1);

	int* num_Elem = (int*)malloc(how_many*sizeof(int));
	f_hamiltonian* d_H = (f_hamiltonian*)malloc(how_many*sizeof(f_hamiltonian));

	int stride[how_many];

	int** basis_Position = (int**)malloc(how_many*sizeof(int*));
	int** basis = (int**)malloc(how_many*sizeof(int*));

	int** d_basis_Position = (int**)malloc(how_many*sizeof(int*));
	int** d_basis = (int**)malloc(how_many*sizeof(int*));

	int** d_Bond = (int**)malloc(how_many*sizeof(int*));

	int padded_dim[how_many];
	int raw_size[how_many];

	dim3* bpg = (dim3*)malloc(how_many*sizeof(dim3));
	dim3* tpb = (dim3*)malloc(how_many*sizeof(dim3));

	hipStream_t stream[how_many];

	hipError_t status[how_many];

	int* d_num_Elem;
	hipMalloc(&d_num_Elem, how_many*sizeof(int));

	for(int i = 0; i<how_many; i++){
		num_Elem[i] = 0;
		stride[i] = 4*lattice_Size[i] + 1;

		d_H[i].fulldim = 2;
		for (int ch=1; ch<lattice_Size[i]; ch++) d_H[i].fulldim *= 2;
		
		basis_Position[i] = (int*)malloc(d_H[i].fulldim*sizeof(int));
		basis[i] = (int*)malloc(d_H[i].fulldim*sizeof(int));

		d_H[i].sectordim = GetBasis(d_H[i].fulldim, lattice_Size[i], Sz[i], basis_Position[i], basis[i]);

		status[i] = hipMalloc(&d_basis_Position[i], d_H[i].fulldim*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error allocating "<<i<<"th d_basis_Position array: "<<hipGetErrorString(status[i])<<endl;
		}	

		status[i] = hipMalloc(&d_basis[i], d_H[i].sectordim*sizeof(int));

		if (status[i] != hipSuccess){
			cout<<"Error allocating "<<i<<"th d_basis array: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipStreamCreate(&stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error creating "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

		num_Elem[i] = d_H[i].sectordim;
		status[i] = hipMemcpy(d_num_Elem, num_Elem, how_many*sizeof(int), hipMemcpyHostToDevice);	
		if (status[i] != hipSuccess){
			cout<<"Error copying num_Elem array to device in "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

	} // can insert more code in here to handle model type later

	for(int i = 0; i<how_many; i++){
		status[i] = hipMemcpyAsync(d_basis_Position[i], basis_Position[i], d_H[i].fulldim*sizeof(int), hipMemcpyHostToDevice, stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error copying "<<i<<"th basis_Position: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipMemcpyAsync(d_basis[i], basis[i], d_H[i].sectordim*sizeof(int), hipMemcpyHostToDevice, stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error copying "<<i<<"th basis: "<<hipGetErrorString(status[i])<<endl;
		}

		padded_dim[i] = (d_H[i].sectordim/1024 + 1)*1024;
		raw_size[i] = (padded_dim[i] + 4*lattice_Size[i]*d_H[i].sectordim);

		status[i] = hipMalloc(&d_H[i].rows, raw_size[i]*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error creating "<<i<<"th rows array: "<<hipGetErrorString(status[i])<<endl;
		}
		status[i] = hipMalloc(&d_H[i].cols, raw_size[i]*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error creating "<<i<<"th cols array: "<<hipGetErrorString(status[i])<<endl;
		}
		status[i] = hipMalloc(&d_H[i].vals, raw_size[i]*sizeof(float));
		if (status[i] != hipSuccess){
			cout<<"Error creating "<<i<<"th values array: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipMalloc(&d_Bond[i], 3*lattice_Size[i]*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error creating "<<i<<"th bonds array: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipMemcpyAsync(d_Bond[i], Bond[i], 3*lattice_Size[i]*sizeof(int), hipMemcpyHostToDevice, stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error copying "<<i<<"th bonds array: "<<hipGetErrorString(status[i])<<endl;
		}
	
		bpg[i].x = (4*lattice_Size[i]*d_H[i].sectordim)/1024 + 1;
		tpb[i].x = 1024;

		status[i] = hipStreamSynchronize(stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error synchronizing "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

		FillDiagonals<<<d_H[i].sectordim/512 + 1, 512, 0, stream[i]>>>(d_basis[i], d_H[i].sectordim, d_H[i].rows, d_H[i].cols, d_H[i].vals, d_Bond[i], lattice_Size[i], JJ[i]);
		
		status[i] = hipStreamSynchronize(stream[i]);

		if (status[i] != hipSuccess){
			cout<<"Error synchronizing "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipPeekAtLastError();
		if (status[i] != hipSuccess){
			cout<<"Error in "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

		
		FillSparse<<<bpg[i].x, tpb[i].x, 0, stream[i]>>>(d_basis_Position[i], d_basis[i], d_H[i].sectordim, d_H[i].rows, d_H[i].cols, d_H[i].vals, d_Bond[i], lattice_Size[i], JJ[i], d_num_Elem, i);

		status[i] = hipPeekAtLastError();
		if (status[i] != hipSuccess){
			cout<<"Error in "<<i<<"th stream: "<<hipGetErrorString(status[i])<<endl;
		}

	}

	/*hamstruct* d_H_sort;
	status2 = hipMalloc(&d_H_sort, *vdim*stride*sizeof(hamstruct));

	if (status2 != hipSuccess){
		std::cout<<"Allocating d_H_sort failed! Error: ";
		std::cout<<hipGetErrorString( status1 )<<std::endl;
		return 1;
	}*/


	hipDeviceSynchronize();

	//int* num_ptr;
	//hipGetSymbolAddress((void**)&num_ptr, (const char*)"d_num_Elem");

	hipMemcpy(num_Elem, d_num_Elem, how_many*sizeof(int), hipMemcpyDeviceToHost);
	//std::cout<<num_Elem<<std::endl;
	for(int i = 0; i < how_many; i++){
      
          status[i] = hipFree(d_basis[i]);
          if ( status[i] != hipSuccess){
            cout<<"Error freeing "<<i<<"th basis array: "<<hipGetErrorString(status[i])<<endl;
          }
	  status[i] = hipFree(d_basis_Position[i]);
	  if (status[i] != hipSuccess){
            cout<<"Error freeing "<<i<<"th basis_Position array: "<<hipGetErrorString(status[i])<<endl;
          }
          status[i] = hipFree(d_Bond[i]); // we don't need these later on
          if (status[i] != hipSuccess){

            cout<<"Error freeing "<<i<<"th Bond array: "<<hipGetErrorString(status[i])<<endl;
          }
        }
	//----------------Sorting Hamiltonian--------------------------//

	
	float** vals_buffer = (float**)malloc(how_many*sizeof(float*));
	int sortnumber[how_many];

	for(int i = 0; i<how_many; i++){
		
		sortEngine_t engine;
		sortStatus_t sortstatus = sortCreateEngine("sort/sort/src/cubin64/", &engine);

		MgpuSortData sortdata;
	
		sortnumber[i];

		sortdata.AttachKey((uint*)d_H[i].rows);
		sortdata.AttachVal(0, (uint*)d_H[i].cols);
		sortdata.AttachVal(1, (uint*)d_H[i].vals);

		sortnumber[i] = ((raw_size[i]/2048) + 1)*2048;

		sortdata.Alloc(engine, sortnumber[i], 2);

		sortdata.firstBit = 0;
		sortdata.endBit = 8*(sizeof(d_H[i].fulldim));

		sortArray(engine, &sortdata);

		/*thrust::device_ptr<int> sort_key_ptr(d_H_rows);
		thrust::device_ptr<int> sort_val_ptr(d_H_cols);

		thrust::sort_by_key(sort_key_ptr, sort_key_ptr + *vdim*stride, sort_val_ptr);*/
        
		status[i] = hipMalloc(&hamil_lancz[i].vals, num_Elem[i]*sizeof(hipDoubleComplex));
		if (status[i] != hipSuccess){
			cout<<"Error allocating "<<i<<"th lancz values array: "<<hipGetErrorString(status[i])<<endl;
		}
		status[i] = hipMalloc(&hamil_lancz[i].rows, num_Elem[i]*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error allocating "<<i<<"th lancz rows array: "<<hipGetErrorString(status[i])<<endl;
		}
		status[i] = hipMalloc(&hamil_lancz[i].cols, num_Elem[i]*sizeof(int));
		if (status[i] != hipSuccess){
			cout<<"Error allocating "<<i<<"th lancz cols array: "<<hipGetErrorString(status[i]);
		}

		hipMemcpy(hamil_lancz[i].rows, (int*)sortdata.keys[0], num_Elem[i]*sizeof(int), hipMemcpyDeviceToDevice);
		
		hipMemcpy(hamil_lancz[i].cols, (int*)sortdata.values1[0], num_Elem[i]*sizeof(int), hipMemcpyDeviceToDevice);
		
		hipMalloc(&vals_buffer[i], num_Elem[i]*sizeof(float));

		hipMemcpy(vals_buffer[i], (float*)sortdata.values2[0], num_Elem[i]*sizeof(float), hipMemcpyDeviceToDevice);
		FullToCOO<<<num_Elem[i]/1024 + 1, 1024>>>(num_Elem[i], vals_buffer[i], hamil_lancz[i].vals, d_H[i].sectordim); // csr and description initializations happen somewhere else

		
		sortReleaseEngine(engine);
		hipFree(d_H[i].rows);
		hipFree(d_H[i].cols);
		hipFree(d_H[i].vals);

		hamil_lancz[i].fulldim = d_H[i].fulldim;
		hamil_lancz[i].sectordim = d_H[i].sectordim;

		hipDoubleComplex* h_vals = (hipDoubleComplex*)malloc(num_Elem[i]*sizeof(hipDoubleComplex));
	  	int* h_rows = (int*)malloc(num_Elem[i]*sizeof(int));
	  	int* h_cols = (int*)malloc(num_Elem[i]*sizeof(int));

		status[i] = hipMemcpy(h_vals, hamil_lancz[i].vals, num_Elem[i]*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
		
		if (status[i] != hipSuccess){
			cout<<"Error copying to h_vals: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipMemcpy(h_rows, hamil_lancz[i].rows, num_Elem[i]*sizeof(int), hipMemcpyDeviceToHost);

		if (status[i] != hipSuccess){
			cout<<"Error copying to h_rows: "<<hipGetErrorString(status[i])<<endl;
		}

		status[i] = hipMemcpy(h_cols, hamil_lancz[i].cols, num_Elem[i]*sizeof(int), hipMemcpyDeviceToHost);

		if (status[i] != hipSuccess){
			cout<<"Error copying to h_cols: "<<hipGetErrorString(status[i])<<endl;
		}

		
		/*if(i == 5){
			ofstream fout;
			fout.open("hamiltonian.log");
				for(int j = 0; j < num_Elem[i]; j++){
					fout<<"("<<h_rows[j]<<","<<h_cols[j]<<")";
					fout<<" - "<<h_vals[j].x<<std::endl;

				}
			fout.close();
		}*/
	
	}


	return num_Elem;
}

__global__ void FillDiagonals(int* d_basis, int dim, int* H_rows, int* H_cols, float* H_vals, int* d_Bond, int lattice_Size, float JJ){

	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int site = threadIdx.x%(lattice_Size);

	unsigned int tempi;

	__shared__ int3 tempbond[16];
	//int3 tempbond[16];

	if (row < dim){
		tempi = d_basis[row];
		(tempbond[site]).x = d_Bond[site];
		(tempbond[site]).y = d_Bond[lattice_Size + site];
		(tempbond[site]).z = d_Bond[2*lattice_Size + site];

		H_vals[row] = HDiagPart(tempi, lattice_Size, tempbond, JJ);
		H_rows[row] = row;
		H_cols[row] = row;

	}

	else {
		H_rows[row] = dim;
	}

}

/* Function FillSparse: this function takes the empty Hamiltonian arrays and fills them up. Each thread in x handles one ket |i>, and each thread in y handles one site T0
Inputs: d_basis_Position - position information about the basis
d_basis - other basis infos
d_dim - the number of kets
H_sort - an array that will store the Hamiltonian
d_Bond - the bond information
d_lattice_Size - the number of lattice sites
JJ - the coupling parameter

*/

__global__ void FillSparse(int* d_basis_Position, int* d_basis, int dim, int* H_rows, int* H_cols, float* H_vals, int* d_Bond, const int lattice_Size, const float JJ, int* num_Elem, int index){

	int ii = (blockDim.x/(2*lattice_Size))*blockIdx.x + threadIdx.x/(2*lattice_Size);
	int T0 = threadIdx.x%(2*lattice_Size);

	#if __CUDA_ARCH__ < 200
		const int array_size = 512;
	#elif __CUDA_ARCH__ >= 200
		const int array_size = 1024;
	#else
       		#error your mom
	#endif

	__shared__ int3 tempbond[16];
	int count;
	__shared__ int temppos[array_size];
	__shared__ float tempval[array_size];
	//__shared__ uint tempi[array_size];
	uint tempi;
	__shared__ uint tempod[array_size];

	int stride = 4*lattice_Size;
	//int tempcount;
	int site = T0%(lattice_Size);
	count = 0;
	int rowtemp;

	int start = (dim/array_size + 1)*array_size;

	int s;
	//int si, sj;//sk,sl; //spin operators
	//unsigned int tempi;// tempod; //tempj;
	//hipDoubleComplex tempD;

	tempi = d_basis[ii];

	__syncthreads();

	bool compare;

	if( ii < dim ){
		if (T0 < 2*lattice_Size){
			//Putting bond info in shared memory
			(tempbond[site]).x = d_Bond[site];
			(tempbond[site]).y = d_Bond[lattice_Size + site];
			(tempbond[site]).z = d_Bond[2*lattice_Size + site];

			__syncthreads();
			//Diagonal Part

			/*temppos[threadIdx.x] = d_basis_Position[tempi[threadIdx.x]];
			tempval[threadIdx.x] = HDiagPart(tempi[threadIdx.x], lattice_Size, tempbond, JJ);

			H_sort[ idx(ii, 0, stride) ].value = tempval[threadIdx.x];
			H_sort[ idx(ii, 0, stride) ].colindex = temppos[threadIdx.x];
			H_sort[ idx(ii, 0, stride) ].rowindex = ii;
			H_sort[ idx(ii, 0, stride) ].dim = dim;*/
                
			//-------------------------------
			//Horizontal bond ---------------
			s = (tempbond[site]).x;
			tempod[threadIdx.x] = tempi;
			tempod[threadIdx.x] ^= (1<<s);
			s = (tempbond[site]).y;
			tempod[threadIdx.x] ^= (1<<s);

			//tempod[threadIdx.x] ^= (1<<si); //toggle bit
			//tempod[threadIdx.x] ^= (1<<sj); //toggle bit

			compare = (d_basis_Position[tempod[threadIdx.x]] > ii);
			temppos[threadIdx.x] = (compare) ? d_basis_Position[tempod[threadIdx.x]] : dim;
			tempval[threadIdx.x] = HOffBondX(site, tempi, JJ);

			count += (int)compare;
			//tempcount = (T0/lattice_Size);
			rowtemp = (T0/lattice_Size) ? ii : temppos[threadIdx.x];			
			rowtemp = (compare) ? rowtemp : dim;

			H_vals[ idx(ii, 4*site + (T0/lattice_Size)+ start, stride) ] = tempval[threadIdx.x]; //(T0/lattice_Size) ? tempval[threadIdx.x] : hipConj(tempval[threadIdx.x]);
			H_cols[ idx(ii, 4*site + (T0/lattice_Size) + start, stride) ] = (T0/lattice_Size) ? temppos[threadIdx.x] : ii;
			H_rows[ idx(ii, 4*site + (T0/lattice_Size) + start, stride) ] = rowtemp;

//Vertical bond -----------------
			s = (tempbond[site]).x;
			tempod[threadIdx.x] = tempi;
			tempod[threadIdx.x] ^= (1<<s);
			s = (tempbond[site]).z;
			tempod[threadIdx.x] ^= (1<<s);

			//tempod[threadIdx.x] ^= (1<<si); //toggle bit
			//tempod[threadIdx.x] ^= (1<<sj); //toggle bit
                 
			compare = (d_basis_Position[tempod[threadIdx.x]] > ii);
			temppos[threadIdx.x] =  (compare) ? d_basis_Position[tempod[threadIdx.x]] : dim;
			tempval[threadIdx.x] = HOffBondY(site,tempi, JJ);

			count += (int)compare;
			//tempcount = (T0/lattice_Size);
			rowtemp = (T0/lattice_Size) ? ii : temppos[threadIdx.x];			
			rowtemp = (compare) ? rowtemp : dim;

			H_vals[ idx(ii, 4*site + 2 + (T0/lattice_Size) + start, stride) ] =  tempval[threadIdx.x]; // (T0/lattice_Size) ? tempval[threadIdx.x] : hipConj(tempval[threadIdx.x]);
			H_cols[ idx(ii, 4*site + 2 + (T0/lattice_Size) + start, stride) ] = (T0/lattice_Size) ? temppos[threadIdx.x] : ii;
			H_rows[ idx(ii, 4*site + 2 + (T0/lattice_Size) + start, stride) ] = rowtemp;
			
			__syncthreads();

			atomicAdd(&num_Elem[index], count);
}
}//end of ii
}//end of FillSparse

/*Function: FullToCOO - takes a full sparse matrix and transforms it into COO format
Inputs - num_Elem - the total number of nonzero elements
H_vals - the Hamiltonian values
H_pos - the Hamiltonian positions
hamil_Values - a 1D array that will store the values for the COO form

*/
__global__ void FullToCOO(int num_Elem, float* H_vals, hipDoubleComplex* hamil_Values, int dim){

	int i = threadIdx.x + blockDim.x*blockIdx.x;

	if (i < num_Elem){

		hamil_Values[i].x = H_vals[i];
		

	}
}
;


